
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hipblas.h>

#define alpha 0.001
#define lambda 0.001

// Debug
__global__ void PrintInt(int *arr, int size);
__global__ void PrintDouble(double *arr, int size);
__global__ void PrintFw(int device, int index, int e);
__global__ void PrintBw(int device, int index, int e);

// Computation
__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double *result, int index, int size);
__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size);
__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_weight, double *delayed_weight, double *next_delta, int W_W, int size, double lr);
__global__ void Sigmoid(double *a, int size);
__global__ void Exponential(double *a, int size);
__global__ void Softmax(double *a, double* sum, int size);
__global__ void CheckCorrect(double *test_result_d, int fw_index, int *max_index, double *num_correct);
__global__ void GetMaxIndex(double *a, int num_node, int *index);
__global__ void GetSum(double *a, int num_node, double *s);
void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i);

// Scheduling
void InputForwardComputation(int index);
void InputForwardCommunication(int index, int e);
void HiddenForwardComputation(int device, int index);
void HiddenForwardCommunication(int device, int index, int e);
void InputForwardBackwardComputation(int fw_index, int bw_index, int e);
void HiddenForwardBackwardComputation(int device, int fw_index, int bw_index, int e);
void HiddenForwardBackwardCommunication(int device, int fw_index, int bw_index, int e);
void OutputForwardComputation(int fw_index);
void OutputForwardBackwardComputation(int fw_index, int bw_index, int e);
void OutputForwardBackwardCommunication(int fw_index, int bw_index, int e);

// Synchronization
__global__ void WaituntilZero(int *ready, int index, int line, int x, int e, int d);
__global__ void WaituntilOne(int *ready, int index, int line, int x, int e, int d);
__global__ void SetFlag(int *ready, int index);

// Build Model
void GetResultAndInput();
void SetLayer(int layer_index);

// Main
void train_model();
void test_accuracy();

struct layer_info {
	double **weight;
	double **a;
	double **a_next;
	double **delta;
	double **delta_next;
	int *is_fw_input_ready;
	int *is_bw_input_ready;
	int *is_fw_output_ready;
	int *is_bw_output_ready;
	int *is_fw_next_input_ready;
	int *is_bw_prev_input_ready;
};

struct layer_info *layer;
hipblasHandle_t *handle;
hipStream_t *stream;

double *input_d, *input_host, *result_d, *result_host, *sum;
double *test_input_d, *test_input_host, *test_result_d, *test_result_host, *d_num_correct, *zero, *d_train_correct;
int *num_node_arr, *cur_fw, *cur_bw, *e, *max_index, *d_max_index;

int num_layer = 0, num_data = 0, epoch = 0, test_num_data = 0;
double num_correct = 0;

clock_t start;

int main() {

	// input model's information
	printf("number of layers : ");
	scanf(" %d", &num_layer);

	num_node_arr = (int *)malloc(sizeof(int) * num_layer);
	printf("number of nodes : ");
	for(int i = 0; i < num_layer; i++)
		scanf(" %d", &num_node_arr[i]);
	for(int i = 0; i < num_layer; i++)
		num_node_arr[i]++;

	printf("number of data : ");
	scanf(" %d", &num_data);

	printf("number of test data : ");
	scanf(" %d", &test_num_data);

	printf("epoch : ");
	scanf(" %d", &epoch);

	// make tool
	handle = (hipblasHandle_t *)malloc(num_layer * sizeof(hipblasHandle_t));
	stream = (hipStream_t *)malloc(2 * num_layer * sizeof(hipStream_t));

	cur_fw = (int *)malloc(num_layer * sizeof(int));
	cur_bw = (int *)malloc(num_layer * sizeof(int));
	for(int i = 0; i < num_layer; i++)
		cur_fw[i] = cur_bw[i] = 0;

	e = (int *)malloc(num_layer * sizeof(int));
	for(int i = 0; i < num_layer; i++)
		e[i] = 0;
	
	zero = (double *)malloc(sizeof(double));
	zero[0] = 0.0;

	// build model
	GetResultAndInput();
	layer = (struct layer_info *)malloc(num_layer * sizeof(struct layer_info));
	for(int i = 0; i < num_layer; i++)
		SetLayer(i);

	// enable peer access
	for(int i = 0; i < num_layer - 1; i++)
	{
		hipSetDevice(i);
		hipDeviceEnablePeerAccess(i + 1, 0);
	}
	for(int i = num_layer - 1; i > 0; i--)
	{
		hipSetDevice(i);
		hipDeviceEnablePeerAccess(i - 1, 0);
	}
	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}
	
	start = clock();
	
	// train and test
	train_model();
	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	printf("%fs\n", (double)(clock() - start)/CLOCKS_PER_SEC);

	// test accuracy
	test_accuracy();

	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	printf("finished\n");

	return 0;
}

void SetLayer(int layer_index)
{
	int cur_node = num_node_arr[layer_index], next_node, num_buffer = (num_layer - layer_index) * 2 - 1;
	double *weight_host;

	hipSetDevice(layer_index);
	hipStreamCreate(&stream[2 * layer_index]);
	hipStreamCreate(&stream[2 * layer_index + 1]);
	hipblasCreate(&handle[layer_index]);
	hipblasSetStream(handle[layer_index], stream[2 * layer_index]);

    layer[layer_index].a = (double **)malloc(num_buffer * sizeof(double *));
	for(int i = 0; i < num_buffer; i++)
        hipMalloc((void**) &layer[layer_index].a[i], num_node_arr[layer_index] * sizeof(double));
    hipMalloc((void**) &layer[layer_index].is_fw_input_ready, num_buffer * sizeof(int));

    // except input layer
    if(layer_index != 0)
    {
        hipMalloc((void**) &layer[layer_index].is_bw_output_ready, num_buffer * sizeof(int));
        hipMalloc((void**) &layer[layer_index].is_bw_prev_input_ready, (num_buffer + 2) * sizeof(int));

        layer[layer_index].delta = (double **)malloc(num_buffer * sizeof(double *));
		for(int i = 0; i < num_buffer; i++)
			hipMalloc((void**) &layer[layer_index].delta[i], num_node_arr[layer_index] * sizeof(double));
    }

	// except output layer
	if(layer_index < num_layer - 1)
	{
        next_node = num_node_arr[layer_index + 1];
        hipMalloc((void**) &layer[layer_index].is_bw_input_ready, num_buffer * sizeof(int));
        hipMalloc((void**) &layer[layer_index].is_fw_output_ready, num_buffer * sizeof(int));
        hipMalloc((void**) &layer[layer_index].is_fw_next_input_ready, (num_buffer - 2) * sizeof(int));

		layer[layer_index].a_next = (double **)malloc(num_buffer * sizeof(double *));
		for(int i = 0; i < num_buffer; i++)
			hipMalloc((void**) &layer[layer_index].a_next[i], next_node * sizeof(double));

		layer[layer_index].delta_next = (double **)malloc(num_buffer * sizeof(double *));
		for(int i = 0; i < num_buffer; i++)
			hipMalloc((void**) &layer[layer_index].delta_next[i], next_node * sizeof(double));

        weight_host = (double *)malloc(cur_node * next_node * sizeof(double));
        layer[layer_index].weight = (double **)malloc(num_buffer * sizeof(double *));
		for (int j = 0; j < cur_node * next_node; j++)
            weight_host[j] = sqrt(6.0 / (cur_node + next_node)) * (rand() / (double)RAND_MAX * 2.0 - 1.0);
        for(int i = 0; i < num_buffer; i++)
        {
            hipMalloc((void**) &layer[layer_index].weight[i], cur_node * next_node * sizeof(double));
            hipMemcpy(layer[layer_index].weight[i], weight_host, cur_node * next_node * sizeof(double), hipMemcpyHostToDevice);
        }
		hipDeviceSynchronize();
		free(weight_host);
	}
}

void GetResultAndInput()
{
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[0] * 3], *p;

	pFile = fopen("mnist_train.csv", "r");
	result_host = (double *)malloc(num_data * sizeof(double));
	input_host = (double *)malloc(num_data * num_node_arr[0] * sizeof(double));
	if(pFile != NULL)
    {   
		for(int r_index = 0, i_index = 0; r_index < num_data;)
		{
			fgets(str_tmp, num_node_arr[0] * 3, pFile);

			// set result
			p = strtok(str_tmp, ",");
			if(atoi(p) == 0)
				result_host[r_index++] = num_node_arr[num_layer - 1] - 1;
			else
				result_host[r_index++] = atoi(p);

			// set input
			input_host[i_index++] = 1.0;
			for (int i = 1; i < num_node_arr[0]; i++)
			{
				p = strtok(NULL, ",");
				input_host[i_index++] = atof(p) / 255.0;
			}
        }       
	}
	hipSetDevice(num_layer - 1);
	hipMalloc((void**) &result_d, num_data * sizeof(double));
	hipMemcpy(result_d, result_host, num_data * sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void**) &sum, sizeof(double));

	hipMalloc((void**) &d_num_correct, sizeof(double));
	hipMalloc((void**) &d_train_correct, sizeof(double));
	hipMalloc((void**) &d_max_index, sizeof(int)); max_index = (int *)malloc(sizeof(int));

	hipSetDevice(0);
	hipMalloc((void**) &input_d, num_data * num_node_arr[0] * sizeof(double));
	hipMemcpy(input_d, input_host, num_data * num_node_arr[0] * sizeof(double), hipMemcpyHostToDevice);
}

void train_model() 
{
	//start stage
	for(int i = 0; i < num_layer - 1; i++)
	{
		if(i == 0)
		{
			while(cur_fw[i] < 2 * (num_layer - 1))
			{
				// printf("%d %d\n", i, cur_fw[i]);
				InputForwardComputation(cur_fw[i]);
				InputForwardCommunication(cur_fw[i]++, 0);
			}
		}
		else
		{
			while(cur_fw[i] < 2 * (num_layer  - 1 - i))
			{
				// printf("%d %d\n", i, cur_fw[i]);
				HiddenForwardComputation(i, cur_fw[i]);
				HiddenForwardCommunication(i, cur_fw[i]++, 0);
			}
		}
	}
	for(int i = 1; i < num_layer; i++)
	{
		if(i == num_layer - 1)
		{
			while(cur_fw[i] < i)
			{
				// printf("%d\t%d, %d\n", i, cur_fw[i], cur_bw[i]);
				OutputForwardBackwardComputation(cur_fw[i], cur_bw[i], 0);
				OutputForwardBackwardCommunication(cur_fw[i]++, cur_bw[i]++, 0);
			}
		}
		else
		{
			while(cur_fw[i] < i + 2 * (num_layer  - 1 - i))
			{
				// printf("%d\t%d, %d\n", i, cur_fw[i], cur_bw[i]);
				HiddenForwardBackwardComputation(i, cur_fw[i], cur_bw[i], 0);
				HiddenForwardBackwardCommunication(i, cur_fw[i]++, cur_bw[i]++, 0);
			}
		}
	}
	
	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}
	
	// steady stage		if (data 개수 홀수) : epoch 홀수 / 짝수일 때 시작 buffer remain different buffer
	while(e[0] < epoch)
	{
		for(int i = 0; i < num_layer; i++)
		{
			if(i == 0)
			{
				InputForwardBackwardComputation(cur_fw[i], cur_bw[i]++, e[i]);
				InputForwardCommunication(cur_fw[i]++, e[i]);
			}
			else if(i == num_layer - 1)
			{
				OutputForwardBackwardComputation(cur_fw[i], cur_bw[i], e[i]);
				OutputForwardBackwardCommunication(cur_fw[i]++, cur_bw[i]++, e[i]);
			}
			else
			{
				HiddenForwardBackwardComputation(i, cur_fw[i], cur_bw[i], e[i]);
				HiddenForwardBackwardCommunication(i, cur_fw[i]++, cur_bw[i]++, e[i]);
			}
		}

		for(int i = 0; i < num_layer; i++)
		{
			if(cur_fw[i] == num_data)
				cur_fw[i] = 0;
			if(cur_bw[i] == num_data)
			{
				cur_bw[i] = 0;
				e[i]++;
			}
		}
	}

	// end stage
	// while(cur_bw[0] < num_data)
	// {
	// 	for(int i = 0; i < num_layer; i++)
	// 	{
	// 		if(i == 0)
	// 		{
	// 			InputBackwardComputation(cur_bw[i]++);
	// 		}
	// 		else if(i != num_layer - 1)
	// 		{
	// 			HiddenBackwardComputation(cur_bw[i]);
	// 			HiddenBackwardCommunication(cur_bw[i]++);
	// 		}
	// 	}
	// }
	
//	print device flag
//	for(int i = 0; i < num_layer; i++)
//	{
//		cudaSetDevice(i);
//		PrintInt<<<1, 1>>>(layer[i].is_fw_input_ready, 0, __LINE__);
//		PrintInt<<<1, 1>>>(layer[i].is_fw_input_ready, 1, __LINE__);
//
//		PrintInt<<<1, 1>>>(layer[i].is_bw_input_ready, 0, __LINE__);
//		PrintInt<<<1, 1>>>(layer[i].is_bw_input_ready, 1, __LINE__);
//
//		PrintInt<<<1, 1>>>(layer[i].is_fw_output_ready, 0, __LINE__);
//		PrintInt<<<1, 1>>>(layer[i].is_fw_output_ready, 1, __LINE__);
//
//		PrintInt<<<1, 1>>>(layer[i].is_bw_output_ready, 0, __LINE__);
//		PrintInt<<<1, 1>>>(layer[i].is_bw_output_ready, 1, __LINE__);
//
//		PrintInt<<<1, 1>>>(layer[i].is_fw_next_input_ready, 0, __LINE__);
//		PrintInt<<<1, 1>>>(layer[i].is_fw_next_input_ready, 1, __LINE__);
//
//		PrintInt<<<1, 1>>>(layer[i].is_bw_prev_input_ready, 0, __LINE__);
//		PrintInt<<<1, 1>>>(layer[i].is_bw_prev_input_ready, 1, __LINE__);
//		PrintEnter<<<1, 1>>>();
//
//		cudaDeviceSynchronize();
//	}
}

void test_accuracy() 
{	
	// Set test data
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[0] * 3], *p;

	pFile = fopen("mnist_test.csv", "r");
	if(pFile != NULL)
	{
		for(int r_index = 0, i_index = 0; r_index < test_num_data;)
		{
			fgets(str_tmp, num_node_arr[0] * 3, pFile);

			// set test_result
			p = strtok(str_tmp, ",");
			if(atoi(p) == 0)
				result_host[r_index++] = num_node_arr[num_layer - 1] - 1;
			else
				result_host[r_index++] = atoi(p);

			// set test_input
			input_host[i_index++] = 1.0;
			for (int i = 1; i < num_node_arr[0]; i++)
			{
				p = strtok(NULL, ",");
				input_host[i_index++] = atof(p) / 255.0;
			}
	    }
	}

	hipSetDevice(0);
	hipMemcpy(input_d, input_host, test_num_data * num_node_arr[0] * sizeof(double), hipMemcpyHostToDevice);

	hipSetDevice(num_layer - 1);
	hipMemcpy(result_d, result_host, test_num_data * sizeof(double), hipMemcpyHostToDevice);

	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	// reset flags
	for(int i = 0 ; i < num_layer - 1; i++)
		cur_fw[i] = i;
	cur_fw[num_layer - 1] = 0;

	
	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		for(int j = 0; j < (num_layer - j) * 2 - 1; j++)
		{
			hipMemset(&layer[i].is_fw_input_ready[j], 0, sizeof(int));

			if(i != num_layer - 1)
			{
				hipMemset(&layer[i].is_fw_output_ready[j], 0, sizeof(int));
				hipMemset(&layer[i].is_bw_input_ready[j], 0, sizeof(int));
			}

			if(i != 0)
				hipMemset(&layer[i].is_bw_output_ready[j], 0, sizeof(int));
		}

		if(i != num_layer - 1)
			for(int j = 0; j < (num_layer - j) * 2 - 3; j++)
				hipMemset(&layer[i].is_fw_next_input_ready[j], 0, sizeof(int));

		if(i != 0)
			for(int j = 0; j < (num_layer - j) * 2 + 1; j++)
				hipMemset(&layer[i].is_bw_prev_input_ready[j], 0, sizeof(int));
	}

	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	while(cur_fw[num_layer - 1] < test_num_data)
	{
		for(int i = 0; i < num_layer; i++)
		{
			if(i == 0)
			{
				InputForwardComputation(cur_fw[i]);
				InputForwardCommunication(cur_fw[i]++, -1);
			}
			else if(i == num_layer - 1)
				OutputForwardComputation(cur_fw[i]++);
			else
			{
				HiddenForwardComputation(i, cur_fw[i]);
				HiddenForwardCommunication(i, cur_fw[i], -1);
			}
		}
	}

	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	hipMemcpy(&num_correct, d_num_correct, sizeof(double), hipMemcpyDeviceToHost);
	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	printf("%lf%%\n", num_correct / test_num_data * 100);
}

void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i)
{
	const double alp = 1.0f;
	const double bet  = 0.0f;
		
	hipblasDgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, B_W, A_H, A_W, &alp, d_B, B_W, d_A, A_W, &bet, d_C, B_W);
}

void InputForwardComputation(int index)
{
    int num_buffer = num_layer * 2 - 1;
    int buffer = index % num_buffer;
    hipSetDevice(0);

    // wait for current layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[0]>>>(layer[0].is_fw_input_ready, buffer, __LINE__, index, 0, 0);
	
	// copy input data
	hipMemcpyAsync(layer[0].a[buffer], input_d + index * num_node_arr[0], num_node_arr[0] * sizeof(double), hipMemcpyDeviceToDevice, stream[0]);
	
	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[0]>>>(layer[0].is_fw_output_ready, buffer, __LINE__, index, 0, 0);
	
	// compute activation
	MatrixMultiply(layer[0].a[buffer], layer[0].weight[buffer], layer[0].a_next[buffer], 1, num_node_arr[0], num_node_arr[1], 0);
	Sigmoid<<<(num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a_next[buffer], num_node_arr[1]);
	
	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[0].is_fw_output_ready[buffer], 1, sizeof(int), stream[0]);
}

void InputForwardCommunication(int index, int e)
{
    int num_buffer = num_layer * 2 - 1;
	int buffer = index % num_buffer;
	int next_buffer;
	if(e == -1)
		next_buffer = 1;
	else
    	next_buffer = index % (num_buffer - 2);
	hipSetDevice(0);

	// wait for current layer's forward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[1]>>>(layer[0].is_fw_output_ready, buffer, __LINE__, index, e, 0);
	
	// wait for next layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[1]>>>(layer[0].is_fw_next_input_ready, next_buffer, __LINE__, index, e, 0);

	// copy foward output (activation) to next layer
	hipMemcpyPeerAsync(layer[1].a[next_buffer], 1, layer[0].a_next[buffer], 0, num_node_arr[1] * sizeof(double), stream[1]);
	
	// current layer's forward output buffer is empty
	hipMemsetAsync(&layer[0].is_fw_output_ready[buffer], 0, sizeof(int), stream[1]);

	// next layer's forward input buffer is full
	hipMemsetAsync(&layer[0].is_fw_next_input_ready[next_buffer], 1, sizeof(int), stream[1]);
	hipMemcpyPeerAsync(&layer[1].is_fw_input_ready[next_buffer], 1, &layer[0].is_fw_next_input_ready[next_buffer], 0, sizeof(int), stream[1]);
}

void InputForwardBackwardComputation(int fw_index, int bw_index, int e)
{
    int num_buffer = num_layer * 2 - 1;
    int fw_buffer = fw_index % num_buffer;
    int bw_buffer = bw_index % num_buffer;
    hipSetDevice(0);

    // wait for current layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[0]>>>(layer[0].is_fw_input_ready, fw_buffer, __LINE__, fw_index, 0, 0);

	// copy input data
	hipMemcpyAsync(layer[0].a[fw_buffer], input_d + fw_index * num_node_arr[0], num_node_arr[0] * sizeof(double), hipMemcpyDeviceToDevice, stream[0]);
	
	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[0]>>>(layer[0].is_fw_output_ready, fw_buffer, __LINE__, fw_index, e, 0);
	
	// compute activation
	MatrixMultiply(layer[0].a[fw_buffer], layer[0].weight[fw_buffer], layer[0].a_next[fw_buffer], 1, num_node_arr[0], num_node_arr[1], 0);
	Sigmoid<<<(num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a_next[fw_buffer], num_node_arr[1]);
	
	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[0].is_fw_output_ready[fw_buffer], 1, sizeof(int), stream[0]);

	// wait for current layer's backward input buffer is full
    WaituntilOne<<<1, 1, 0, stream[0]>>>(layer[0].is_bw_input_ready, bw_buffer, __LINE__, bw_index, e, 0);

	// update weight
    UpdateWeight<<<(num_node_arr[0] * num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a[bw_buffer], layer[0].weight[fw_buffer], layer[0].weight[(fw_buffer + 1) % num_buffer], layer[0].weight[bw_buffer], layer[0].delta_next[bw_buffer], num_node_arr[1], num_node_arr[1] * num_node_arr[0], alpha);

    // current layer's forward input buffer is empty
    hipMemsetAsync(&layer[0].is_fw_input_ready[bw_buffer], 0, sizeof(int), stream[0]);

	// current layer's backward input buffer is empty
	hipMemsetAsync(&layer[0].is_bw_input_ready[bw_buffer], 0, sizeof(int), stream[0]);
	hipMemcpyPeerAsync(&layer[1].is_bw_prev_input_ready[bw_buffer], 1, &layer[0].is_bw_input_ready[bw_buffer], 0, sizeof(int), stream[0]);
}

void HiddenForwardComputation(int device, int index)
{
	int num_buffer = (num_layer - device) * 2 - 1;
    int buffer = index % num_buffer;
	hipSetDevice(device);

	// wait for current layer's forward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_input_ready, buffer, __LINE__, index, 0, device);

	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_output_ready, buffer, __LINE__, index, 0, device);

	// compute activation
	MatrixMultiply(layer[device].a[buffer], layer[device].weight[buffer], layer[device].a_next[buffer], 1, num_node_arr[device], num_node_arr[device + 1], device);
	if(device != num_layer - 2)
		Sigmoid<<<(num_node_arr[device + 1] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].a_next[buffer], num_node_arr[device + 1]);

	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[device].is_fw_output_ready[buffer], 1, sizeof(int), stream[2 * device]);

	// current layer's forward input buffer is empty
	hipMemsetAsync(&layer[device].is_fw_input_ready[buffer], 0, sizeof(int), stream[2 * device]);
	hipMemcpyPeerAsync(&layer[device - 1].is_fw_next_input_ready[buffer], device - 1, &layer[device].is_fw_input_ready[buffer], device, sizeof(int), stream[2 * device]);
}

void HiddenForwardCommunication(int device, int index, int e)
{
	int num_buffer = (num_layer - device) * 2 - 1;
	int buffer = index % num_buffer;
	int next_buffer;
	if(e == -1)
		next_buffer = (index + 1) % (num_buffer- 2);
	else
		next_buffer = index % (num_buffer - 2);
	hipSetDevice(device);
	
	// wait for current layer's forward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_output_ready, buffer, __LINE__, index, 0, device);
				
	// wait for next layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_next_input_ready, next_buffer, __LINE__, index, 0, device);

	// copy forward output (activation) to next layer
	hipMemcpyPeerAsync(layer[device + 1].a[next_buffer], device + 1, layer[device].a_next[buffer], device, num_node_arr[device + 1] * sizeof(double), stream[2 * device + 1]);

	// current layer's forward output buffer is empty
	hipMemsetAsync(&layer[device].is_fw_output_ready[buffer], 0, sizeof(int), stream[2 * device + 1]);

	// next layer's forward input buffer is full
	hipMemsetAsync(&layer[device].is_fw_next_input_ready[next_buffer], 1, sizeof(int), stream[2 * device + 1]);
	hipMemcpyPeerAsync(&layer[device + 1].is_fw_input_ready[next_buffer], device + 1, &layer[device].is_fw_next_input_ready[next_buffer], device, sizeof(int), stream[2 * device + 1]);
}

void HiddenForwardBackwardComputation(int device, int fw_index, int bw_index, int e)
{
	int num_buffer = (num_layer - device) * 2 - 1;
    int fw_buffer = fw_index % num_buffer;
	int bw_buffer = bw_index % num_buffer;

	hipSetDevice(device);

	// wait for current layer's forward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_input_ready, fw_buffer, __LINE__, fw_index, e, device);

	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_output_ready, fw_buffer, __LINE__, fw_index, e, device);

	// compute activation
	MatrixMultiply(layer[device].a[fw_buffer], layer[device].weight[fw_buffer], layer[device].a_next[fw_buffer], 1, num_node_arr[device], num_node_arr[device + 1], device);
	if(device != num_layer - 2)
		Sigmoid<<<(num_node_arr[device + 1] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].a_next[fw_buffer], num_node_arr[device + 1]);

	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[device].is_fw_output_ready[fw_buffer], 1, sizeof(int), stream[2 * device]);

	// wait for current layer's backward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_bw_input_ready, bw_buffer, __LINE__, bw_index, e, device);

	// wait for current layer's backward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_bw_output_ready, bw_buffer, __LINE__, bw_index, e, device);

	// compute backward output (delta)
	MatrixMultiply(layer[device].weight[bw_buffer], layer[device].delta_next[bw_buffer], layer[device].delta[bw_buffer], num_node_arr[device], num_node_arr[device + 1], 1, device);
	GetHiddenLayerDelta<<<(num_node_arr[device] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].delta[bw_buffer], layer[device].a[bw_buffer], layer[device].weight[bw_buffer], layer[device].delta_next[bw_buffer], num_node_arr[device]);

	// current layer's backward output buffer is full
	hipMemsetAsync(&layer[device].is_bw_output_ready[bw_buffer], 1, sizeof(int), stream[2 * device]);

	// update weight
	UpdateWeight<<<(num_node_arr[device] * num_node_arr[device + 1] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].a[bw_buffer], layer[device].weight[fw_buffer], layer[device].weight[(fw_buffer + 1) % num_buffer], layer[device].weight[bw_buffer], layer[device].delta_next[bw_buffer], num_node_arr[device + 1], num_node_arr[device + 1] * num_node_arr[device], alpha);

	// current layer's forward/backward input buffer is empty
	hipMemsetAsync(&layer[device].is_fw_input_ready[fw_buffer], 0, sizeof(int), stream[2 * device]);
	hipMemsetAsync(&layer[device].is_bw_input_ready[bw_buffer], 0, sizeof(int), stream[2 * device]);
	hipMemcpyPeerAsync(&layer[device - 1].is_fw_next_input_ready[fw_buffer], device - 1, &layer[device].is_fw_input_ready[fw_buffer], device, sizeof(int), stream[2 * device]);
	hipMemcpyPeerAsync(&layer[device + 1].is_bw_prev_input_ready[bw_buffer], device + 1, &layer[device].is_bw_input_ready[bw_buffer], device, sizeof(int), stream[2 * device]);
}

void HiddenForwardBackwardCommunication(int device, int fw_index, int bw_index, int e)
{
	int num_buffer = (num_layer - device) * 2 - 1;
	int fw_buffer = fw_index % num_buffer;
	int next_buffer = fw_index % (num_buffer - 2);
	int bw_buffer = bw_index % num_buffer;
	int prev_buffer = bw_index % (num_buffer + 2);
	hipSetDevice(device);

	// wait for current layer's forward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_output_ready, fw_buffer, __LINE__, fw_index, e, device);
				
	// wait for next layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_next_input_ready, next_buffer, __LINE__, fw_index, e, device);

	// copy forward output (activation) to next layer
	hipMemcpyPeerAsync(layer[device + 1].a[next_buffer], device + 1, layer[device].a_next[fw_buffer], device, num_node_arr[device + 1] * sizeof(double), stream[2 * device + 1]);

	// current layer's forward output buffer is empty
	hipMemsetAsync(&layer[device].is_fw_output_ready[fw_buffer], 0, sizeof(int), stream[2 * device + 1]);

	// next layer's forward input buffer is full
	hipMemsetAsync(&layer[device].is_fw_next_input_ready[next_buffer], 1, sizeof(int), stream[2 * device + 1]);
	hipMemcpyPeerAsync(&layer[device + 1].is_fw_input_ready[next_buffer], device + 1, &layer[device].is_fw_next_input_ready[next_buffer], device, sizeof(int), stream[2 * device + 1]);
	//cudaMemsetAsync(&layer[device + 1].is_fw_input_ready[next_buffer], 1, sizeof(int), stream[2 * device + 1]);

	// wait for current layer's backward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_bw_output_ready, bw_buffer, __LINE__, bw_index, e, device);

	// wait for previous layer's backward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_bw_prev_input_ready, prev_buffer, __LINE__, bw_index, e, device);

	// copy backward output (delta) to preious layer
	hipMemcpyPeerAsync(layer[device - 1].delta_next[prev_buffer], device - 1, layer[device].delta[bw_buffer], device, num_node_arr[device - 1] * sizeof(double), stream[2 * device + 1]);

	// current layer's backward output buffer is empty
	hipMemsetAsync(&layer[device].is_bw_output_ready[bw_buffer], 0, sizeof(int), stream[2 * device + 1]);

	// previous layer's backward input buffer is full
	hipMemsetAsync(&layer[device].is_bw_prev_input_ready[prev_buffer], 1, sizeof(int), stream[2 * device + 1]);
	hipMemcpyPeerAsync(&layer[device - 1].is_bw_input_ready[prev_buffer], device - 1, &layer[device].is_bw_prev_input_ready[prev_buffer], device, sizeof(int), stream[2 * device + 1]);
}

void OutputForwardComputation(int fw_index)
{
	int fw_buffer = 0;
	hipSetDevice(num_layer - 1);

	// wait for current layer's forward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].is_fw_input_ready, fw_buffer, __LINE__, fw_index, 0, num_layer - 1);

	// get estimated class
	GetMaxIndex<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[fw_buffer], num_node_arr[num_layer - 1], d_max_index);

	// compare with real class
	CheckCorrect<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(result_d, fw_index, d_max_index, d_num_correct);

	// current layer's forward input buffer is empty
	hipMemsetAsync(&layer[num_layer - 1].is_fw_input_ready[fw_buffer], 0, sizeof(int), stream[2 * (num_layer - 1)]);
	hipMemcpyPeerAsync(&layer[num_layer - 2].is_fw_next_input_ready[fw_buffer], num_layer - 2, &layer[num_layer - 1].is_fw_input_ready[fw_buffer], num_layer - 1, sizeof(int), stream[2 * (num_layer - 1)]);
}

void OutputForwardBackwardComputation(int fw_index, int bw_index, int e)
{
	int fw_buffer = 0;
	int bw_buffer = 0;
	hipSetDevice(num_layer - 1);

	// wait for current layer's forward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].is_fw_input_ready, fw_buffer, __LINE__, fw_index, e, num_layer - 1);

	// softmax
	Exponential<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[fw_buffer], num_node_arr[num_layer - 1]);
	GetSum<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[fw_buffer], num_node_arr[num_layer - 1], sum);
	Softmax<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[fw_buffer], sum, num_node_arr[num_layer - 1]);

	// get estimated class
	GetMaxIndex<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[fw_buffer], num_node_arr[num_layer - 1], d_max_index);

	// compare with real class
	CheckCorrect<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(result_d, fw_index, d_max_index, d_train_correct);

	// wait for current layer's backward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].is_bw_output_ready, bw_buffer, __LINE__, bw_index, e, num_layer - 1);
		
	// compute delta
	GetOutputLayerDelta<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[bw_buffer], layer[num_layer - 1].delta[bw_buffer], result_d, bw_index, num_node_arr[num_layer - 1]);

	// current layer's backward output buffer is full
	hipMemsetAsync(&layer[num_layer - 1].is_bw_output_ready[bw_buffer], 1, sizeof(int), stream[2 * (num_layer - 1)]);

	// current layer's forward input buffer is empty
	hipMemsetAsync(&layer[num_layer - 1].is_fw_input_ready[fw_buffer], 0, sizeof(int), stream[2 * (num_layer - 1)]);
	hipMemcpyPeerAsync(&layer[num_layer - 2].is_fw_next_input_ready[fw_buffer], num_layer - 2, &layer[num_layer - 1].is_fw_input_ready[fw_buffer], num_layer - 1, sizeof(int), stream[2 * (num_layer - 1)]);
}

void OutputForwardBackwardCommunication(int fw_index, int bw_index, int e)
{
	int bw_buffer = 0;
	int prev_buffer = bw_index % 3;
	hipSetDevice(num_layer - 1);

	// wait for previous layer's backward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * num_layer - 1]>>>(layer[num_layer - 1].is_bw_prev_input_ready, prev_buffer, __LINE__, bw_index, e, num_layer - 1);

	// wait for current layer's backward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * num_layer - 1]>>>(layer[num_layer - 1].is_bw_output_ready, bw_buffer, __LINE__, bw_index, e, num_layer - 1);

	// copy backward output (delta) to previous layer
	hipMemcpyPeerAsync(layer[num_layer - 2].delta_next[prev_buffer], num_layer - 2, layer[num_layer - 1].delta[bw_buffer], num_layer - 1, num_node_arr[num_layer - 1] * sizeof(double), stream[2 * num_layer -1]);

	// current layer's backward output buffer is empty
	hipMemsetAsync(&layer[num_layer - 1].is_bw_output_ready[bw_buffer], 0, sizeof(int), stream[2 * num_layer - 1]);

	// previous layer's backward input buffer is full
	hipMemsetAsync(&layer[num_layer - 1].is_bw_prev_input_ready[prev_buffer], 1, sizeof(int), stream[2 * num_layer - 1]);
	hipMemcpyPeerAsync(&layer[num_layer - 2].is_bw_input_ready[prev_buffer], num_layer - 2, &layer[num_layer - 1].is_bw_prev_input_ready[prev_buffer], num_layer - 1, sizeof(int), stream[2 * num_layer - 1]);
}

__global__ void GetSum(double *a, int num_node, double *s)
{
	s[0] = 0.0;
	for(int i = 1; i < num_node; i++)
		s[0] += a[i];
}

__global__ void GetMaxIndex(double *a, int num_node, int *index)
{
	double max = a[1];
	index[0] = 1;
	printf("%f ", a[1]);
	for(int i = 2; i < num_node; i++)
	{
		printf("%f ", a[i]);
		if(a[i] > max)
		{
			max = a[i];
			index[0] = i;
		}
	}
	printf("\n", index[0]);
}
__global__ void CheckCorrect(double *test_result_d, int fw_index, int *max_index, double *num_correct)
{
	printf("%d %d\n", (int)test_result_d[fw_index] ,max_index[0]);
	if((int)test_result_d[fw_index] == max_index[0])
			num_correct[0]++;
}

__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double *result, int index, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
	{
		if(i != result[index])
			output_delta[i] = 0.0 - output_a[i];
		else
			output_delta[i] = 1.0 - output_a[i];
	}
}

__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		cur_delta[i] = (1.0 - cur_a[i]) * cur_a[i] * (cur_delta[i] - cur_weight[i] * prev_delta[0]);
}

__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_weight, double *delayed_weight, double *next_delta, int W_W, int size, double lr)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double DC;
    if(i < size)
    {
        DC = cur_a[i / W_W] * next_delta[i % W_W];
        next_weight[i] = cur_weight[i] + lr * (DC + 0.001 * DC * DC * (cur_weight[i] - delayed_weight[i]));
    }
}

__global__ void Sigmoid(double *a, int size) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = 1.0 / (1.0 + exp(-a[i]));
	if(i == 0)
		a[i] = 1.0;
}

__global__ void Exponential(double *a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = exp(a[i]);
	if(i == 0)
		a[i] = 0.0;
}

__global__ void Softmax(double *a, double* sum, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] /= sum[0];
}

__global__ void WaituntilZero(int *ready, int index, int line, int x, int e, int d)
{
	while(ready[index] != 0)
	{
		// printf("line: %d\tindex: %d\tepoch: %d\tdevice: %d\n", line, x, e, d);
	}
}

__global__ void WaituntilOne(int *ready, int index, int line, int x, int e, int d)
{
	while(ready[index] == 0)
	{
		// printf("line: %d\tindex: %d\tepoch: %d\tdevice: %d\n", line, x, e, d);
	}
}

__global__ void SetFlag(int *ready, int index)
{
	ready[index] = 1 - ready[index];
}

__global__ void PrintFw(int device, int index, int e)
{
	printf("device #%d\t fw -> %d\tepoch : %d\n", device, index, e);
}

__global__ void PrintBw(int device, int index, int e)
{
	printf("device #%d\t bw -> %d\tepoch : %d\n", device, index, e);
}

__global__ void PrintInt(int *arr, int size)
{
	for(int i = 0; i < size; i++)
		printf("%d ", arr[i]);
	printf("\n");
}

__global__ void PrintDouble(double *arr, int size)
{
	for(int i = 0; i < size; i++)
		printf("%lf ", arr[i]);
	printf("\n");
}
