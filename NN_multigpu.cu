#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hipblas.h>


#define alpha 0.001

__global__ void SetResult(double *result_d, int correct_result, int size);
__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double result, int size);
__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size);
__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_delta, int W_W, int size, double lr);
__global__ void Print(double *a, int size);
__global__ void Printint(int *a, int size);
__global__ void Sigmoid(double *a, int size);
__global__ void Exponential(double *a, int size);
__global__ void Softmax(double *a, double sum, int size);
__global__ void ForwardWait(int *ready, int device, int index);
__global__ void BackwardWait(int *ready, int device, int index);
__global__ void SetBackwardNotReady(int *ready, int index);
__global__ void SetForwardNotReady(int *ready, int index);
__global__ void PrintBw(int device, int index);
__global__ void PrintFw(int device, int index);
void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i);
void SetLayer(int layer_index);
void train_model();
void test_accuracy();

struct layer_info {
	double **a;
	double **weight;
	double **delta;
	double **delta_next;
	double **a_next;
	int *one;
	int *fw_ready;
	int *bw_ready;
};

struct layer_info *layer;
hipblasHandle_t *handle;
hipStream_t *stream;

double *input_d, *input_host;
int *result_d, *result_host, *num_node_arr, *cur_fw, *cur_bw;

int num_layer = 1, num_data, epoch;

clock_t start;


int main() {

	// input model's information
	printf("number of layers : ");
	scanf(" %d", &num_layer);

	num_node_arr = (int *)malloc(sizeof(int) * num_layer);
	printf("number of nodes : ");
	for(int i = 0; i < num_layer; i++)
		scanf(" %d", &num_node_arr[i]);
	for(int i = 0; i < num_layer; i++)
		num_node_arr[i]++;

	printf("number of data : ");
	scanf(" %d", &num_data);
	
	// make tool
	handle = (hipblasHandle_t *)malloc(num_layer * sizeof(hipblasHandle_t));
	stream = (hipStream_t *)malloc(num_layer * sizeof(hipStream_t));

	// build model
	layer = (struct layer_info *)malloc(num_layer * sizeof(struct layer_info));
	for(int i = 0; i < num_layer; i++)
		SetLayer(i);
	cur_fw = (int *)malloc(num_layer * sizeof(int));
	cur_bw = (int *)malloc(num_layer * sizeof(int));
	for(int i = 0; i < num_layer; i++)
		cur_fw[i] = cur_bw[i] = 0;

	hipDeviceSynchronize();

	// enable peer access
	for(int i = 0; i < num_layer - 1; i++)
	{
		hipSetDevice(i);
		hipDeviceEnablePeerAccess(i + 1, 0);
	}
	
	// train and test
	printf("epoch : ");
	scanf(" %d", &epoch);
	train_model();

	return 0;
}

void SetLayer(int layer_index)
{
	int cur_node = num_node_arr[layer_index], next_node, *one_h;
	double *weight_host;

	hipSetDevice(layer_index);
	hipStreamCreate(&stream[layer_index]);
	hipblasCreate(&handle[layer_index]);
	hipblasSetStream(handle[layer_index], stream[layer_index]);

	hipMalloc((void**) &layer[layer_index].fw_ready, num_layer * sizeof(int));
	hipMalloc((void**) &layer[layer_index].bw_ready, num_layer * sizeof(int));
	hipMalloc((void**) &layer[layer_index].one, sizeof(int));
	one_h = (int *)malloc(sizeof(int));
	one_h[0] = 1;
	hipMemcpy(layer[layer_index].one, one_h, sizeof(int), hipMemcpyHostToDevice);


	layer[layer_index].a = (double **)malloc(num_layer * sizeof(double *));
	for(int i = 0; i < num_layer; i++)
		hipMalloc((void**) &layer[layer_index].a[i], num_node_arr[layer_index] * sizeof(double));

	// except input layer
	if(layer_index != 0)
	{
		layer[layer_index].delta = (double **)malloc(num_layer * sizeof(double *));
		for(int i = 0; i < num_layer; i++)
			hipMalloc((void**) &layer[layer_index].delta[i], num_node_arr[layer_index] * sizeof(double));
	}
	// except output layer
	if(layer_index < num_layer - 1)
	{
		next_node = num_node_arr[layer_index + 1];
		layer[layer_index].a_next = (double **)malloc(num_layer * sizeof(double *));
		layer[layer_index].delta_next = (double **)malloc(num_layer * sizeof(double *));
		for(int i = 0; i < num_layer; i++)
		{
			hipMalloc((void**) &layer[layer_index].a_next[i], next_node * sizeof(double));
			hipMalloc((void**) &layer[layer_index].delta_next[i], next_node * sizeof(double));
		}
		layer[layer_index].weight = (double **)malloc(num_layer * sizeof(double *));
		for(int i = 0; i < num_layer; i++)
		{
			weight_host = (double *)malloc(cur_node * next_node * sizeof(double));
			for (int j = 0; j < cur_node * next_node; j++)
				weight_host[j] = sqrt(6.0 / (cur_node + next_node)) * (rand() / (double)RAND_MAX * 2.0 - 1.0);
			hipMalloc((void**) &layer[layer_index].weight[i], cur_node * next_node * sizeof(double));
			hipMemcpy(layer[layer_index].weight[i], weight_host, cur_node * next_node * sizeof(double), hipMemcpyHostToDevice);
			free(weight_host);
		}
	}
}


void train_model() 
{
	int result_index = 0, input_index = 0;
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[0] * 3], *p;
	double sum;

	pFile = fopen("mnist_train.csv", "r");
	result_host = (int *)malloc(num_data * sizeof(int));
	input_host = (double *)malloc(num_data * num_node_arr[0] * sizeof(double));
	if(pFile != NULL)
    {   
		while(1)
		{
			fgets(str_tmp, num_node_arr[0] * 5, pFile);
			if (feof(pFile))
				break;
			
			// set result
			p = strtok(str_tmp, ",");
			if(atoi(p) == 0)
				result_host[result_index++] = num_node_arr[num_layer - 1] - 1;
			else
				result_host[result_index++] = atoi(p);

			// set input
			input_host[input_index++] = 1.0;
			for (int i = 1; i < num_node_arr[0]; i++)
			{
				p = strtok(NULL, ",");
				input_host[input_index++] = atof(p) / 255.0;
			}
        }       
	}
	hipSetDevice(num_layer - 1);
	hipMalloc((void**) &result_d, num_data * sizeof(int));
	hipMemcpy(result_d, result_host, num_data * sizeof(int), hipMemcpyHostToDevice);
	hipSetDevice(0);
	hipMalloc((void**) &input_d, num_data * num_node_arr[0] * sizeof(double));
	hipMemcpy(input_d, input_host, num_data * num_node_arr[0] * sizeof(double), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	result_index = input_index = 0;

	// startup stage
	// input layer
	hipSetDevice(0);
	for(int j = 0; j < num_layer; j++)
	{	
		hipMemcpyAsync(layer[0].a[j], input_d + result_index++ * num_node_arr[0], num_node_arr[0] * sizeof(double), hipMemcpyDeviceToDevice, stream[0]);
		MatrixMultiply(layer[0].a[j], layer[0].weight[j], layer[0].a_next[j], 1, num_node_arr[0], num_node_arr[1], 0);
		Sigmoid<<<(num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a_next[j], num_node_arr[1]);
		hipMemcpyPeerAsync(layer[1].a[j], 1, layer[0].a_next[j], 0, num_node_arr[1] * sizeof(double), stream[0]);
		hipMemcpyPeerAsync(layer[1].fw_ready + j, 1, layer[0].one , 0, sizeof(int), stream[0]);
		PrintFw<<<1, 1, 0, stream[0]>>>(0, j);
	} // Forward  X num_layer
	hipDeviceSynchronize();
	BackwardWait<<<1, 1, 0, stream[0]>>>(layer[0].bw_ready, 0, 0);
	UpdateWeight<<<(num_node_arr[0] * num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a[0], layer[0].weight[0], layer[0].delta_next[0], num_node_arr[1], num_node_arr[1] * num_node_arr[0], alpha);
	PrintBw<<<1, 1, 0, stream[0]>>>(0, 0);
	SetBackwardNotReady<<<1, 1, 0, stream[0]>>>(layer[0].bw_ready, 0);

	hipSetDevice(1);
	Printint<<<1, 1, 0, stream[1]>>>(layer[1].fw_ready, num_layer);
	hipStreamSynchronize(stream[1]);

	hipSetDevice(0);
	hipStreamSynchronize(stream[0]);
	
	/*
	// hidden layer
	for(int i = 1; i < num_layer - 1; i++)
	{
		cudaSetDevice(i);
		for(int j = 0; j < num_layer; j++)
		{
			Printint<<<1, 1, 0, stream[i]>>>(layer[i].fw_ready, num_layer);
			ForwardWait<<<1, 1, 0, stream[i]>>>(layer[i].fw_ready, i, j);
			MatrixMultiply(layer[i].a[j], layer[i].weight[j], layer[i].a_next[j], 1, num_node_arr[i], num_node_arr[i + 1], i);
			if(i != num_layer - 2)
				Sigmoid<<<(num_node_arr[i + 1] + 1023) / 1024, 1024, 0, stream[i]>>>(layer[i].a_next[j], num_node_arr[i + 1]);
			
			cudaMemcpyPeerAsync(layer[i + 1].a[j], i + 1, layer[i].a_next[j], i, num_node_arr[i + 1] * sizeof(double), stream[i]);
			cudaMemcpyPeerAsync(layer[i + 1].fw_ready + j, i + 1, layer[i].one , i, sizeof(int), stream[i]);
			SetForwardNotReady<<<1, 1, 0, stream[i]>>>(layer[i].fw_ready, j);
			PrintFw<<<1, 1, 0, stream[i]>>>(i, j);
		}
		for(int j = 0; j < i + 1; j++)
		{
			BackwardWait<<<1, 1, 0, stream[i]>>>(layer[i].bw_ready, i, j);
			MatrixMultiply(layer[i].weight[j], layer[i].delta_next[j], layer[i].delta[j], num_node_arr[i], num_node_arr[i + 1], 1, i);
			GetHiddenLayerDelta<<<(num_node_arr[i] + 1023) / 1024, 1024>>>(layer[i].delta[j], layer[i].a[j], layer[i].weight[j], layer[i].delta_next[j], num_node_arr[i]);
			cudaMemcpyPeerAsync(layer[i - 1].delta_next[j], i - 1, layer[i].delta[j], i, num_node_arr[i] * sizeof(double), stream[i]);
			cudaMemcpyPeerAsync(layer[i - 1].bw_ready + j, i - 1, layer[i].one , i, sizeof(int), stream[0]);
			SetBackwardNotReady<<<1, 1, 0, stream[i]>>>(layer[i].bw_ready, j);
			PrintBw<<<1, 1, 0, stream[i]>>>(i, j);
		}
	}

	
	// output layer
	cudaSetDevice(num_layer - 1);
	for(int j = 0; j < num_layer; j++)
	{
		ForwardWait<<<1, 1, 0, stream[num_layer - 1]>>>(layer[num_layer - 1].fw_ready, num_layer - 1, j);
		Exponential<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024>>>(layer[num_layer - 1].a[j], num_node_arr[num_layer - 1]);
		cublasDasum(handle[num_layer - 1], num_node_arr[num_layer - 1], layer[num_layer - 1].a[j], 1, &sum);
		Softmax<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024>>>(layer[num_layer - 1].a[j], sum, num_node_arr[num_layer - 1]);
		PrintFw<<<1, 1, 0, stream[num_layer - 1]>>>(num_layer - 1, j);
		PrintBw<<<1, 1, 0, stream[num_layer - 1]>>>(num_layer - 1, j);
		GetOutputLayerDelta<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024>>>(layer[num_layer - 1].a[j], layer[num_layer - 1].delta[j], result_d[j], num_node_arr[num_layer - 1]);
		cudaMemcpyPeerAsync(layer[num_layer - 2].delta_next[j], num_layer - 2, layer[num_layer - 1].delta[j], num_layer - 1, num_node_arr[num_layer - 1] * sizeof(double), stream[num_layer - 1]);
		SetForwardNotReady<<<1, 1, 0, stream[num_layer - 1]>>>(layer[num_layer - 1].fw_ready, j);
	}

	for(int i=0; i< num_layer; i++)
		cudaStreamSynchronize(stream[i]);
		*/

	/*
	// steady stage
	// input layer
	Setinput
	Forward
	BackwardWait
	Backward
	SetBackwardNotReady

	// hidden layer
	for(int i = 1; i < num_layer - 2; i++)
	{
		cudaSetDevice(i);
		ForwardWait<<<1, 1, stream[i]>>>(forward_ready[i], cur_fw[i]);
		Forward
		SetForwardNotReady
		BackwardWait
		Backward
		SetBackwardNotReady
	}

	// output layer
	ForwardWait<<<1, 1, stream[i]>>>(forward_ready[i], cur_fw[i]);
	Forward
	SetForwardNotReady
	GetOutputdelta
	Backward
	*/

	return; 
}

__global__ void ForwardWait(int *ready, int device, int index)
{
	while(ready[index] == 0) {
		//printf("device : %d , fw: %d\n", device, index);
	}
}

__global__ void BackwardWait(int *ready, int device, int index)
{
	while(ready[index] == 0) {
		//printf("device : %d , bw: %d\n", device, index);
	}
}

__global__ void SetForwardNotReady(int *ready, int index)
{
	ready[index] = 0;
}

__global__ void SetBackwardNotReady(int *ready, int index)
{
	ready[index] = 0;
}

/*
void test_accuracy() 
{	
	double correct = 0, num_test_examples = 0, result_index = 0, sum;
	int max_index;
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[1] * 3];
	char* p;

	pFile = fopen("mnist_test.csv", "r");

	if (pFile != NULL)
	{
		while (1)
		{
			// load data from file
			fgets(str_tmp, num_node_arr[1] * 3, pFile);
			if (feof(pFile))
				break;
			p = strtok(str_tmp, ",");

			// set result
			if(atoi(p) == 0)
				result_index = num_node_arr[num_layer] - 1;
			else
				result_index = atof(p);

			// set input
			p = strtok(NULL, ",");
			input_host[0] = 1.0;
			for (int i = 1; i < num_node_arr[1]; i++)
			{
				input_host[i] = atof(p) / 255.0;
				p = strtok(NULL, ",");
			}
			cudaMemcpy(layer[1].a, input_host, num_node_arr[1] * sizeof(double), cudaMemcpyHostToDevice);

			// forward pass
			for(int i = 1; i < num_layer - 1; i++)
			{
				MatrixMultiply(layer[i].weight, layer[i].a, layer[i+1].a, num_node_arr[i+1], num_node_arr[i], 1);
				Sigmoid<<<(num_node_arr[i+1] + 1023) / 1024, 1024>>>(layer[i+1].a, num_node_arr[i+1]);
			}
			MatrixMultiply(layer[num_layer - 1].weight, layer[num_layer - 1].a, layer[num_layer].a, num_node_arr[num_layer], num_node_arr[num_layer - 1], 1);
			Exponential<<<(num_node_arr[num_layer] + 1023) / 1024, 1024>>>(layer[num_layer].a, num_node_arr[num_layer]);
			cublasDasum(handle, num_node_arr[num_layer], layer[num_layer].a, 1, &sum);
			Softmax<<<(num_node_arr[num_layer] + 1023) / 1024, 1024>>>(layer[num_layer].a, sum, num_node_arr[num_layer]);

			//Print<<<1, 1>>>(layer[num_layer].a, num_node_arr[num_layer]);
			//cudaDeviceSynchronize();
			
			cublasIdamax(handle, num_node_arr[num_layer], layer[num_layer].a, 1, &max_index);

			if (result_index == --max_index)
				correct++;

			num_test_examples++;
		}
	}

	if(pFile != NULL)
		fclose(pFile);
	
	printf("%lf%%\n", correct / num_test_examples * 100);

	return;
}*/

void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i)
{
	const double alp = 1.0f;
	const double bet  = 0.0f;
		
	hipblasDgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, B_W, A_H, A_W, &alp, d_B, B_W, d_A, A_W, &bet, d_C, B_W);
}


__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double result, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
	{
		if(i != result)
			output_delta[i] = 0.0 - output_a[i];
		else
			output_delta[i] = 1.0 - output_a[i];
	}
}

__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		cur_delta[i] = (1.0 - cur_a[i]) * cur_a[i] * (cur_delta[i] - cur_weight[i] * prev_delta[0]);
}


__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_delta, int W_W, int size, double lr)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		cur_weight[i] = cur_weight[i] + lr * cur_a[i / W_W] * next_delta[i % W_W];
}

__global__ void Sigmoid(double *a, int size) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = 1.0 / (1.0 + exp(-a[i]));
	if(i == 0)
		a[i] = 1.0;
}


__global__ void Exponential(double *a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = exp(a[i]);
	if(i == 0)
		a[i] = 0.0;
}

__global__ void Softmax(double *a, double sum, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] /= sum;
}


__global__ void Print(double *a, int size)
{
	for(int i=0; i<size; i++)
		printf(" %lf /", a[i]);
	printf("\n");
}

__global__ void Printint(int *a, int size)
{
	for(int i=0; i<size; i++)
		printf(" %d /", a[i]);
	printf("\n");
}

__global__ void PrintFw(int device, int index)
{
	printf("device : %d , fw : %d\n", device, index);
}

__global__ void PrintBw(int device, int index)
{
	printf("device : %d , fw : %d\n", device, index);
}


