#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>
#include "hip/hip_runtime_api.h"
#include <omp.h>

#define alpha 0.001

__global__ void SetResult(double *result_d, int correct_result, int size);
__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double *result, int size);
__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size);
__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_delta, int W_W, int size, double lr);
__global__ void Print(double *a, int size);
__global__ void Printint(int *a, int device, int size);
__global__ void Sigmoid(double *a, int size);
__global__ void Exponential(double *a, int size);
__global__ void Softmax(double *a, double sum, int size);
__global__ void PrintBw(int device, int index);
__global__ void PrintFw(int device, int index);
void* StartupStageOutputLayer(void *arg);
void* StartupStageHiddenLayer(void* index);
void* StartupStageInputLayer(void *arg);
void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i);
void SetLayer(int layer_index);
void train_model();
void test_accuracy();

struct layer_info {
	double **a;
	double **weight;
	double **delta;
	double **delta_next;
	double **a_next;
	int *one;
	int *zero;
	int *fw_ready;
	int *bw_ready;
};

struct layer_info *layer;
hipblasHandle_t *handle;
hipStream_t *stream;
pthread_t *thread;

double *input_d, *input_host, *result_d, *result_host;
int *num_node_arr, *cur_fw, *cur_bw;

int num_layer = 1, num_data, epoch, thr_id;

clock_t start;


int main() {

	// input model's information
	printf("number of layers : ");
	scanf(" %d", &num_layer);

	num_node_arr = (int *)malloc(sizeof(int) * num_layer);
	printf("number of nodes : ");
	for(int i = 0; i < num_layer; i++)
		scanf(" %d", &num_node_arr[i]);
	for(int i = 0; i < num_layer; i++)
		num_node_arr[i]++;

	printf("number of data : ");
	scanf(" %d", &num_data);
	
	// make tool
	handle = (hipblasHandle_t *)malloc(num_layer * sizeof(hipblasHandle_t));
	stream = (hipStream_t *)malloc(num_layer * sizeof(hipStream_t));
	thread = (pthread_t *)malloc(num_layer * sizeof(pthread_t));

	// build model
	layer = (struct layer_info *)malloc(num_layer * sizeof(struct layer_info));
	for(int i = 0; i < num_layer; i++)
		SetLayer(i);
	cur_fw = (int *)malloc(num_layer * sizeof(int));
	cur_bw = (int *)malloc(num_layer * sizeof(int));
	for(int i = 0; i < num_layer; i++)
		cur_fw[i] = cur_bw[i] = 0;

	hipDeviceSynchronize();

	// enable peer access
	for(int i = 0; i < num_layer - 1; i++)
	{
		hipSetDevice(i);
		hipDeviceEnablePeerAccess(i + 1, 0);
	}
	
	// train and test
	printf("epoch : ");
	scanf(" %d", &epoch);
	train_model();

	return 0;
}

void SetLayer(int layer_index)
{
	int cur_node = num_node_arr[layer_index], next_node;
	double *weight_host;

	hipSetDevice(layer_index);
	hipStreamCreate(&stream[layer_index]);
	hipblasCreate(&handle[layer_index]);
	hipblasSetStream(handle[layer_index], stream[layer_index]);

	layer[layer_index].fw_ready = (int *)malloc(num_layer * sizeof(int));
	layer[layer_index].bw_ready = (int *)malloc(num_layer * sizeof(int));
	for(int i = 0; i < num_layer; i++)
		layer[layer_index].fw_ready[i] = layer[layer_index].bw_ready[i] = 0;

	layer[layer_index].a = (double **)malloc(num_layer * sizeof(double *));
	for(int i = 0; i < num_layer; i++)
		hipMalloc((void**) &layer[layer_index].a[i], num_node_arr[layer_index] * sizeof(double));

	// except input layer
	if(layer_index != 0)
	{
		layer[layer_index].delta = (double **)malloc(num_layer * sizeof(double *));
		for(int i = 0; i < num_layer; i++)
			hipMalloc((void**) &layer[layer_index].delta[i], num_node_arr[layer_index] * sizeof(double));
	}
	// except output layer
	if(layer_index < num_layer - 1)
	{
		next_node = num_node_arr[layer_index + 1];
		layer[layer_index].a_next = (double **)malloc(num_layer * sizeof(double *));
		layer[layer_index].delta_next = (double **)malloc(num_layer * sizeof(double *));
		for(int i = 0; i < num_layer; i++)
		{
			hipMalloc((void**) &layer[layer_index].a_next[i], next_node * sizeof(double));
			hipMalloc((void**) &layer[layer_index].delta_next[i], next_node * sizeof(double));
		}
		layer[layer_index].weight = (double **)malloc(num_layer * sizeof(double *));
		for(int i = 0; i < num_layer; i++)
		{
			weight_host = (double *)malloc(cur_node * next_node * sizeof(double));
			for (int j = 0; j < cur_node * next_node; j++)
				weight_host[j] = sqrt(6.0 / (cur_node + next_node)) * (rand() / (double)RAND_MAX * 2.0 - 1.0);
			hipMalloc((void**) &layer[layer_index].weight[i], cur_node * next_node * sizeof(double));
			hipMemcpy(layer[layer_index].weight[i], weight_host, cur_node * next_node * sizeof(double), hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			free(weight_host);
		}
	}
}


void train_model() 
{
	int result_index = 0, input_index = 0, status;
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[0] * 3], *p;
	int t1, t2 = 1;

	pFile = fopen("mnist_train.csv", "r");
	result_host = (double *)malloc(num_data * sizeof(double));
	input_host = (double *)malloc(num_data * num_node_arr[0] * sizeof(double));
	if(pFile != NULL)
    {   
		while(1)
		{
			fgets(str_tmp, num_node_arr[0] * 5, pFile);
			if (feof(pFile))
				break;
			
			// set result
			p = strtok(str_tmp, ",");
			if(atoi(p) == 0)
				result_host[result_index++] = num_node_arr[num_layer - 1] - 1;
			else
				result_host[result_index++] = atoi(p);

			// set input
			input_host[input_index++] = 1.0;
			for (int i = 1; i < num_node_arr[0]; i++)
			{
				p = strtok(NULL, ",");
				input_host[input_index++] = atof(p) / 255.0;
			}
        }       
	}
	hipSetDevice(num_layer - 1);
	hipMalloc((void**) &result_d, num_data * sizeof(int));
	hipMemcpy(result_d, result_host, num_data * sizeof(int), hipMemcpyHostToDevice);

	hipSetDevice(0);
	hipMalloc((void**) &input_d, num_data * num_node_arr[0] * sizeof(double));
	hipMemcpy(input_d, input_host, num_data * num_node_arr[0] * sizeof(double), hipMemcpyHostToDevice);

	result_index = input_index = 0;

	// startup stage
	thr_id = pthread_create(&thread[0], NULL, StartupStageInputLayer, NULL);
	// StartupStageInputLayer(NULL);
	// thr_id = pthread_create(&thread[1], NULL, StartupStageHiddenLayer, (void *)&t2);

	// for(int i = 1; i < num_layer - 1; i++)
	// 	thr_id = pthread_create(&thread[i], NULL, StartupStageHiddenLayer, (void *)&t); 	

	// for(int i = 0; i < num_layer ; i++)
	// 	pthread_join(thread[i], (void **) &status);
	/*
	// steady stage
	// input layer
	Setinput
	Forward
	BackwardWait
	Backward
	SetBackwardNotReady

	// hidden layer
	for(int i = 1; i < num_layer - 2; i++)
	{
		hipSetDevice(i);
		ForwardWait<<<1, 1, stream[i]>>>(forward_ready[i], cur_fw[i]);
		Forward
		SetForwardNotReady
		BackwardWait
		Backward
		SetBackwardNotReady
	}

	// output layer
	ForwardWait<<<1, 1, stream[i]>>>(forward_ready[i], cur_fw[i]);
	Forward
	SetForwardNotReady
	GetOutputdelta
	Backward
	*/

	return; 
}

void* StartupStageInputLayer(void *arg)
{
	checkCudaErrors(hipSetDevice(0));
	PrintFw<<<1, 1>>>(0, 0);
	hipDeviceSynchronize();
	// PrintFw<<<1, 1, 0, stream[0]>>>(0, 0);
	// hipStreamSynchronize(stream[0]);
	// for(int j = 0; j < num_layer; j++)
	// {	
	// 	hipMemcpyAsync(layer[0].a[j], input_d + j * num_node_arr[0], num_node_arr[0] * sizeof(double), hipMemcpyDeviceToDevice, stream[0]); // 
	// 	MatrixMultiply(layer[0].a[j], layer[0].weight[j], layer[0].a_next[j], 1, num_node_arr[0], num_node_arr[1], 0);
	// 	Sigmoid<<<(num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a_next[j], num_node_arr[1]);

	// 	while(layer[1].fw_ready[j] == 1) { }

	// 	hipMemcpyPeerAsync(layer[1].a[j], 1, layer[0].a_next[j], 0, num_node_arr[1] * sizeof(double), stream[0]);
	// 	hipStreamSynchronize(stream[0]);
	// 	layer[1].fw_ready[j] = 1;
		
	// 	PrintFw<<<1, 1, 0, stream[0]>>>(0, j);
	// 	hipStreamSynchronize(stream[0]);
	// } // Forward  X num_layer

	// while(layer[0].bw_ready[0] == 0) { }

	// UpdateWeight<<<(num_node_arr[0] * num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a[0], layer[0].weight[0], layer[0].delta_next[0], num_node_arr[1], num_node_arr[1] * num_node_arr[0], alpha);

	// hipStreamSynchronize(stream[0]);
	// layer[0].bw_ready[0] = 0;

	// PrintBw<<<1, 1, 0, stream[0]>>>(0, 0);
	// hipStreamSynchronize(stream[0]);
}

void* StartupStageHiddenLayer(void* index)
{
	int i = *((int *)index);
	hipSetDevice(i);
	for(int j = 0; j < num_layer; j++)
	{
		printf("OOOOOO\n");
		Printint<<<1, 1, 0, stream[i]>>>(layer[i].fw_ready, i, num_layer);
		hipStreamSynchronize(stream[i]);

		printf("OOOOOO\n");
		while(layer[i].fw_ready[j] == 0) { }

		printf("OOOOOO\n");
		MatrixMultiply(layer[i].a[j], layer[i].weight[j], layer[i].a_next[j], 1, num_node_arr[i], num_node_arr[i + 1], i);
		if(i != num_layer - 2)
			Sigmoid<<<(num_node_arr[i + 1] + 1023) / 1024, 1024, 0, stream[i]>>>(layer[i].a_next[j], num_node_arr[i + 1]);
		
		printf("OOOOOO\n");
		while(layer[i + 1].fw_ready[j] == 1) { }
		hipMemcpyPeerAsync(layer[i + 1].a[j], i + 1, layer[i].a_next[j], i, num_node_arr[i + 1] * sizeof(double), stream[i]);
		hipMemcpyAsync(layer[i].fw_ready + j, layer[i].zero, sizeof(int), hipMemcpyDeviceToHost, stream[i]);
		hipMemcpyAsync(layer[i + 1].fw_ready + j, layer[i].one , sizeof(int), hipMemcpyDeviceToHost, stream[i]);

		printf("OOOOOO\n");
		PrintFw<<<1, 1, 0, stream[i]>>>(i, j);
		hipStreamSynchronize(stream[i]);
	}
	
	for(int j = 0; j < i + 1; j++)
	{
		Printint<<<1, 1, 0, stream[i]>>>(layer[i].bw_ready, i, num_layer);
		hipStreamSynchronize(stream[i]);

		while(layer[i].bw_ready[j] == 0) { }

		MatrixMultiply(layer[i].weight[j], layer[i].delta_next[j], layer[i].delta[j], num_node_arr[i], num_node_arr[i + 1], 1, i);
		GetHiddenLayerDelta<<<(num_node_arr[i] + 1023) / 1024, 1024, 0, stream[i]>>>(layer[i].delta[j], layer[i].a[j], layer[i].weight[j], layer[i].delta_next[j], num_node_arr[i]);

		while(layer[i - 1].bw_ready[j] == 1) { }
		hipMemcpyPeerAsync(layer[i - 1].delta_next[j], i - 1, layer[i].delta[j], i, num_node_arr[i] * sizeof(double), stream[i]);
		hipMemcpyAsync(layer[i].bw_ready + j, layer[i].zero, sizeof(int), hipMemcpyDeviceToHost, stream[i]);
		hipMemcpyAsync(layer[i - 1].bw_ready + j, layer[i].one , sizeof(int), hipMemcpyDeviceToHost, stream[i]);

		PrintBw<<<1, 1, 0, stream[i]>>>(i, j);
		hipStreamSynchronize(stream[i]);
	}
}

void* StartupStageOutputLayer(void *arg)
{
	double sum;
	hipSetDevice(num_layer - 1);
	for(int j = 0; j < num_layer; j++)
	{
		Printint<<<1, 1, 0, stream[num_layer - 1]>>>(layer[num_layer - 1].fw_ready, num_layer - 1, num_layer);
		hipStreamSynchronize(stream[num_layer - 1]);

		while(layer[num_layer - 1].fw_ready[j] == 0) { }

		Exponential<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[num_layer - 1]>>>(layer[num_layer - 1].a[j], num_node_arr[num_layer - 1]);
		hipblasDasum(handle[num_layer - 1], num_node_arr[num_layer - 1], layer[num_layer - 1].a[j], 1, &sum);
		Softmax<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[num_layer - 1]>>>(layer[num_layer - 1].a[j], sum, num_node_arr[num_layer - 1]);

		PrintFw<<<1, 1, 0, stream[num_layer - 1]>>>(num_layer - 1, j);
		hipStreamSynchronize(stream[num_layer - 1]);

		GetOutputLayerDelta<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[num_layer - 1]>>>(layer[num_layer - 1].a[j], layer[num_layer - 1].delta[j], result_d + j, num_node_arr[num_layer - 1]);

		PrintBw<<<1, 1, 0, stream[num_layer - 1]>>>(num_layer - 1, j);
		hipStreamSynchronize(stream[num_layer - 1]);

		while(layer[num_layer - 2].bw_ready[j] == 1) { }
		hipMemcpyPeerAsync(layer[num_layer - 2].delta_next[j], num_layer - 2, layer[num_layer - 1].delta[j], num_layer - 1, num_node_arr[num_layer - 1] * sizeof(double), stream[num_layer - 1]);
		hipMemcpyAsync(layer[num_layer - 1].fw_ready + j, layer[num_layer - 1].zero, sizeof(int), hipMemcpyDeviceToHost, stream[num_layer - 1]);
		hipMemcpyAsync(layer[num_layer - 2].bw_ready + j, layer[num_layer - 1].one , sizeof(int), hipMemcpyDeviceToHost, stream[num_layer - 1]);
	}
}

/*
void test_accuracy() 
{	
	double correct = 0, num_test_examples = 0, result_index = 0, sum;
	int max_index;
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[1] * 3];
	char* p;

	pFile = fopen("mnist_test.csv", "r");

	if (pFile != NULL)
	{
		while (1)
		{
			// load data from file
			fgets(str_tmp, num_node_arr[1] * 3, pFile);
			if (feof(pFile))
				break;
			p = strtok(str_tmp, ",");

			// set result
			if(atoi(p) == 0)
				result_index = num_node_arr[num_layer] - 1;
			else
				result_index = atof(p);

			// set input
			p = strtok(NULL, ",");
			input_host[0] = 1.0;
			for (int i = 1; i < num_node_arr[1]; i++)
			{
				input_host[i] = atof(p) / 255.0;
				p = strtok(NULL, ",");
			}
			hipMemcpy(layer[1].a, input_host, num_node_arr[1] * sizeof(double), hipMemcpyHostToDevice);

			// forward pass
			for(int i = 1; i < num_layer - 1; i++)
			{
				MatrixMultiply(layer[i].weight, layer[i].a, layer[i+1].a, num_node_arr[i+1], num_node_arr[i], 1);
				Sigmoid<<<(num_node_arr[i+1] + 1023) / 1024, 1024>>>(layer[i+1].a, num_node_arr[i+1]);
			}
			MatrixMultiply(layer[num_layer - 1].weight, layer[num_layer - 1].a, layer[num_layer].a, num_node_arr[num_layer], num_node_arr[num_layer - 1], 1);
			Exponential<<<(num_node_arr[num_layer] + 1023) / 1024, 1024>>>(layer[num_layer].a, num_node_arr[num_layer]);
			hipblasDasum(handle, num_node_arr[num_layer], layer[num_layer].a, 1, &sum);
			Softmax<<<(num_node_arr[num_layer] + 1023) / 1024, 1024>>>(layer[num_layer].a, sum, num_node_arr[num_layer]);

			//Print<<<1, 1>>>(layer[num_layer].a, num_node_arr[num_layer]);
			//hipDeviceSynchronize();
			
			hipblasIdamax(handle, num_node_arr[num_layer], layer[num_layer].a, 1, &max_index);

			if (result_index == --max_index)
				correct++;

			num_test_examples++;
		}
	}

	if(pFile != NULL)
		fclose(pFile);
	
	printf("%lf%%\n", correct / num_test_examples * 100);

	return;
}*/

void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i)
{
	const double alp = 1.0f;
	const double bet  = 0.0f;
		
	hipblasDgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, B_W, A_H, A_W, &alp, d_B, B_W, d_A, A_W, &bet, d_C, B_W);
}

__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double *result, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
	{
		if(i != result[0])
			output_delta[i] = 0.0 - output_a[i];
		else
			output_delta[i] = 1.0 - output_a[i];
	}
}

__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		cur_delta[i] = (1.0 - cur_a[i]) * cur_a[i] * (cur_delta[i] - cur_weight[i] * prev_delta[0]);
}

__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_delta, int W_W, int size, double lr)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		cur_weight[i] = cur_weight[i] + lr * cur_a[i / W_W] * next_delta[i % W_W];
}

__global__ void Sigmoid(double *a, int size) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = 1.0 / (1.0 + exp(-a[i]));
	if(i == 0)
		a[i] = 1.0;
}

__global__ void Exponential(double *a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = exp(a[i]);
	if(i == 0)
		a[i] = 0.0;
}

__global__ void Softmax(double *a, double sum, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] /= sum;
}

__global__ void Print(double *a, int size)
{
	for(int i=0; i<size; i++)
		printf(" %lf /", a[i]);
	printf("\n");
}

__global__ void Printint(int *a, int device, int size)
{
	printf("device : %d , ", device);
	for(int i=0; i<size; i++)
		printf(" %d /", a[i]);
	printf("\n");
}

__global__ void PrintFw(int device, int index)
{
	printf("device : %d , fw : %d\n", device, index);
}

__global__ void PrintBw(int device, int index)
{
	printf("device : %d , bw : %d\n", device, index);
}


