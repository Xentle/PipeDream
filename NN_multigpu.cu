
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <hipblas.h>

#define alpha 0.001

// Debug
__global__ void PrintInt(int *flag, int index, int line);
__global__ void PrintDouble(double *flag, int index, int size, int line);
__global__ void PrintFw(int device, int index);
__global__ void PrintBw(int device, int index);

// Computation
__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double *result, int index, int size);
__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size);
__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_delta, int W_W, int size, double lr);
__global__ void Sigmoid(double *a, int size);
__global__ void Exponential(double *a, int size);
__global__ void Softmax(double *a, double sum, int size);
void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i);

// Scheduling
void InputForwardComputation(int index);
void InputForwardCommunication(int index);
void HiddenForwardComputation(int device, int index);
void HiddenForwardCommunication(int device, int index);
void InputForwardBackwardComputation(int fw_index, int bw_index);
void InputForwardBackwardCommunication(int fw_index, int bw_index);
void HiddenForwardBackwardComputation(int device, int fw_index, int bw_index);
void HiddenForwardBackwardCommunication(int device, int fw_index, int bw_index);
void OutputForwardBackwardComputation(int fw_index, int bw_index);
void OutputForwardBackwardCommunication(int fw_index, int bw_index);

// Synchronization
__global__ void WaituntilZero(int *ready, int index, int line, int x);
__global__ void WaituntilOne(int *ready, int index, int line, int x);
__global__ void SetFlag(int *ready, int index);

// Build Model
void GetResultAndInput();
void SetLayer(int layer_index);

// Main
void train_model();
void test_accuracy();

struct layer_info {
	double *weight;
	double **a;
	double **a_next;
	double **delta;
	double **delta_next;
	int *is_fw_input_ready;
	int *is_bw_input_ready;
	int *is_fw_output_ready;
	int *is_bw_output_ready;
	int *is_fw_next_input_ready;
	int *is_bw_prev_input_ready;
};

struct layer_info *layer;
hipblasHandle_t *handle;
hipStream_t *stream;

double *input_d, *input_host, *result_d, *result_host, *sum;
int *num_node_arr, *cur_fw, *cur_bw, *e;

int num_layer = 0, num_data = 0, epoch = 0;

clock_t start;

int main() {

	// input model's information
	printf("number of layers : ");
	scanf(" %d", &num_layer);

	num_node_arr = (int *)malloc(sizeof(int) * num_layer);
	printf("number of nodes : ");
	for(int i = 0; i < num_layer; i++)
		scanf(" %d", &num_node_arr[i]);
	for(int i = 0; i < num_layer; i++)
		num_node_arr[i]++;

	printf("number of data : ");
	scanf(" %d", &num_data);

	printf("epoch : ");
	scanf(" %d", &epoch);
	
	// make tool
	handle = (hipblasHandle_t *)malloc(num_layer * sizeof(hipblasHandle_t));
	stream = (hipStream_t *)malloc(2 * num_layer * sizeof(hipStream_t));

	cur_fw = (int *)malloc(num_layer * sizeof(int));
	cur_bw = (int *)malloc(num_layer * sizeof(int));
	for(int i = 0; i < num_layer; i++)
		cur_fw[i] = cur_bw[i] = 0;

	e = (int *)malloc(num_layer * sizeof(int));
	for(int i = 0; i < num_layer; i++)
		e[i] = 0;

	sum = (double *)malloc(2 * sizeof(double));
	sum[0] = sum[1] = 0;

	// build model
	GetResultAndInput();
	layer = (struct layer_info *)malloc(num_layer * sizeof(struct layer_info));
	for(int i = 0; i < num_layer; i++)
		SetLayer(i);

	// enable peer access
	for(int i = 0; i < num_layer - 1; i++)
	{
		hipSetDevice(i);
		hipDeviceEnablePeerAccess(i + 1, 0);
	}
	for(int i = num_layer - 1; i > 0; i--)
	{
		hipSetDevice(i);
		hipDeviceEnablePeerAccess(i - 1, 0);
	}

	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}
	
	// train and test
	train_model();

	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}

	printf("finished\n");

	return 0;
}

void SetLayer(int layer_index)
{
	int cur_node = num_node_arr[layer_index], next_node;
	double *weight_host;

	hipSetDevice(layer_index);
	hipStreamCreate(&stream[2 * layer_index]);
	hipStreamCreate(&stream[2 * layer_index + 1]);
	hipblasCreate(&handle[layer_index]);
	hipblasSetStream(handle[layer_index], stream[2 * layer_index]);

	hipMallocManaged((void**) &layer[layer_index].is_fw_output_ready, 2 * sizeof(int));
	hipMallocManaged((void**) &layer[layer_index].is_bw_output_ready, 2 * sizeof(int));
	layer[layer_index].is_fw_output_ready[0] = layer[layer_index].is_fw_output_ready[1] = 0;
	layer[layer_index].is_bw_output_ready[0] = layer[layer_index].is_bw_output_ready[1] = 0;

	hipMallocManaged((void**) &layer[layer_index].is_fw_input_ready, 2 * sizeof(int));
	hipMallocManaged((void**) &layer[layer_index].is_bw_input_ready, 2 * sizeof(int));
	layer[layer_index].is_fw_input_ready[0] = layer[layer_index].is_fw_input_ready[1] = 0;
	layer[layer_index].is_bw_input_ready[0] = layer[layer_index].is_bw_input_ready[1] = 0;

	hipMallocManaged((void**) &layer[layer_index].is_fw_next_input_ready, 2 * sizeof(int));
	hipMallocManaged((void**) &layer[layer_index].is_bw_prev_input_ready, 2 * sizeof(int));
	layer[layer_index].is_fw_next_input_ready[0] = layer[layer_index].is_fw_next_input_ready[1] = 0;
	layer[layer_index].is_bw_prev_input_ready[0] = layer[layer_index].is_bw_prev_input_ready[1] = 0;
	
	layer[layer_index].a = (double **)malloc(2 * sizeof(double *));
	for(int i = 0; i < 2; i++)
		hipMalloc((void**) &layer[layer_index].a[i], num_node_arr[layer_index] * sizeof(double));

	// except input layer
	if(layer_index != 0)
	{
		layer[layer_index].delta = (double **)malloc(2 * sizeof(double *));
		for(int i = 0; i < 2; i++)
			hipMalloc((void**) &layer[layer_index].delta[i], num_node_arr[layer_index] * sizeof(double));
	}

	// except output layer
	if(layer_index < num_layer - 1)
	{
		next_node = num_node_arr[layer_index + 1];

		layer[layer_index].a_next = (double **)malloc(2 * sizeof(double *));
		for(int i = 0; i < 2; i++)
			hipMalloc((void**) &layer[layer_index].a_next[i], next_node * sizeof(double));

		layer[layer_index].delta_next = (double **)malloc(2 * sizeof(double *));
		for(int i = 0; i < 2; i++)
			hipMalloc((void**) &layer[layer_index].delta_next[i], next_node * sizeof(double));

		weight_host = (double *)malloc(cur_node * next_node * sizeof(double));
		for (int j = 0; j < cur_node * next_node; j++)
			weight_host[j] = sqrt(6.0 / (cur_node + next_node)) * (rand() / (double)RAND_MAX * 2.0 - 1.0);
		hipMalloc((void**) &layer[layer_index].weight, cur_node * next_node * sizeof(double));
		hipMemcpy(layer[layer_index].weight, weight_host, cur_node * next_node * sizeof(double), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		free(weight_host);
	}
}

void GetResultAndInput()
{
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[0] * 3], *p;

	pFile = fopen("mnist_train.csv", "r");
	result_host = (double *)malloc(num_data * sizeof(double));
	input_host = (double *)malloc(num_data * num_node_arr[0] * sizeof(double));
	if(pFile != NULL)
    {   
		for(int r_index = 0, i_index = 0; r_index < num_data;)
		{
			fgets(str_tmp, num_node_arr[0] * 3, pFile);

			// set result
			p = strtok(str_tmp, ",");
			if(atoi(p) == 0)
				result_host[r_index++] = num_node_arr[num_layer - 1] - 1;
			else
				result_host[r_index++] = atoi(p);

			// set input
			input_host[i_index++] = 1.0;
			for (int i = 1; i < num_node_arr[0]; i++)
			{
				p = strtok(NULL, ",");
				input_host[i_index++] = atof(p) / 255.0;
			}
        }       
	}
	hipSetDevice(num_layer - 1);
	hipMalloc((void**) &result_d, num_data * sizeof(int));
	hipMemcpy(result_d, result_host, num_data * sizeof(int), hipMemcpyHostToDevice);

	hipSetDevice(0);
	hipMalloc((void**) &input_d, num_data * num_node_arr[0] * sizeof(double));
	hipMemcpy(input_d, input_host, num_data * num_node_arr[0] * sizeof(double), hipMemcpyHostToDevice);
}

void train_model() 
{
	//start stage
	for(int i = 0; i < num_layer - 1; i++)
	{
		if(i == 0)
		{
			while(cur_fw[i] < 2 * (num_layer - 1))
			{
				InputForwardComputation(cur_fw[i]);
				InputForwardCommunication(cur_fw[i]++);
			}
		}
		else
		{
			while(cur_fw[i] < 2 * (num_layer  - 1 - i))
			{
				HiddenForwardComputation(i, cur_fw[i]);
				HiddenForwardCommunication(i, cur_fw[i]++);
			}
		}
	}

	
	for(int i = 1; i < num_layer; i++)
	{
		if(i == num_layer - 1)
		{
			while(cur_fw[i] < i)
			{
				OutputForwardBackwardComputation(cur_fw[i], cur_bw[i]);
				OutputForwardBackwardCommunication(cur_fw[i]++, cur_bw[i]++);
			}
		}
		else
		{
			while(cur_fw[i] < i + 2 * (num_layer  - 1 - i))
			{
				HiddenForwardBackwardComputation(i, cur_fw[i], cur_bw[i]);
				HiddenForwardBackwardCommunication(i, cur_fw[i]++, cur_bw[i]++);
			}
		}
	}
	
	for(int i = 0; i < num_layer; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}
	
	// steady stage		epoch 홀수 or 짝수 -> cur_Bf = index % 2 or 1 - (index % 2)
	for(int i = 0; i < num_layer; i++)
	{
		if(i == 0)
		{
			while(e[i] < epoch - 1)
			{
				InputForwardBackwardComputation(cur_fw[i], cur_bw[i]++);
				InputForwardCommunication(cur_fw[i]++);
				if(cur_fw[i] == num_data)
					cur_fw[i] = 0;
				if(cur_bw[i] == num_data)
				{
					e[i]++;
					cur_bw[i] = 0;
				}
			}

			// while(cur_bw[i] < num_data - 2)
			// {
			// 	InputForwardBackwardComputation(cur_fw[i], cur_bw[i]++);
			// 	InputForwardCommunication(cur_fw[i]++);
			// 	if(cur_fw[i] == num_data)
			// 		cur_fw[i] = 0;
			// }
		}
		else if(i == num_layer - 1)
		{
			while(e[i] < epoch - 1)
			// while(e[i] < epoch)
			{
				OutputForwardBackwardComputation(cur_fw[i], cur_bw[i]);
				OutputForwardBackwardCommunication(cur_fw[i]++, cur_bw[i]++);
				if(cur_fw[i] == num_data)
					cur_fw[i] = 0;
				if(cur_bw[i] == num_data)
				{
					e[i]++;
					cur_bw[i] = 0;
				}
			}
		}
		else
		{
			while(e[i] < epoch - 1)
			{
				HiddenForwardBackwardComputation(i, cur_fw[i], cur_bw[i]);
				HiddenForwardBackwardCommunication(i, cur_fw[i]++, cur_bw[i]++);
				if(cur_fw[i] == num_data)
					cur_fw[i] = 0;
				if(cur_bw[i] == num_data)
				{
					e[i]++;
					cur_bw[i] = 0;
				}
			}

			// while(cur_bw[i] < num_data - 2)
			// {
			// 	HiddenForwardBackwardComputation(i, cur_fw[i], cur_bw[i]);
			// 	HiddenForwardBackwardCommunication(i, cur_fw[i]++, cur_bw[i]++);
			// 	if(cur_fw[i] == num_data)
			// 		cur_fw[i] = 0;
			// }
		}
	}

	/*
	// end stage
	for(int i = 0; i < num_layer; i++)
	{
		if(i == 0)
		{
			while(cur_bw[i] <= num_data)
			{
				InputBackwardComputation(cur_fw[i], cur_bw[i]);
				InputBackwardCommunication(cur_fw[i]++, cur_bw[i]++);
			}
		}
		else if(i == num_layer - 1)
		{
			while(cur_bw[i] <= num_data)
			{
				OutputBackwardComputation(cur_fw[i], cur_bw[i]);
				OutputBackwardCommunication(cur_fw[i]++, cur_bw[i]++);
			}
		}
		else
		{
			while(cur_bw[i] <= num_data)
			{
				HiddenBackwardComputation(cur_fw[i], cur_bw[i]);
				HiddenBackwardCommunication(cur_fw[i]++, cur_bw[i]++);
			}
		}
	}
	*/
}

/*
void test_accuracy() 
{	
	double correct = 0, num_test_examples = 0, result_index = 0, sum;
	int max_index;
	FILE* pFile = NULL;
	char str_tmp[num_node_arr[1] * 3];
	char* p;
	pFile = fopen("mnist_test.csv", "r");
	if (pFile != NULL)
	{
		while (1)
		{
			// load data from file
			fgets(str_tmp, num_node_arr[1] * 3, pFile);
			if (feof(pFile))
				break;
			p = strtok(str_tmp, ",");
			// set result
			if(atoi(p) == 0)
				result_index = num_node_arr[num_layer] - 1;
			else
				result_index = atof(p);
			// set input
			p = strtok(NULL, ",");
			input_host[0] = 1.0;
			for (int i = 1; i < num_node_arr[1]; i++)
			{
				input_host[i] = atof(p) / 255.0;
				p = strtok(NULL, ",");
			}
			cudaMemcpy(layer[1].a, input_host, num_node_arr[1] * sizeof(double), cudaMemcpyHostToDevice);
			// forward pass
			for(int i = 1; i < num_layer - 1; i++)
			{
				MatrixMultiply(layer[i].weight, layer[i].a, layer[i+1].a, num_node_arr[i+1], num_node_arr[i], 1);
				Sigmoid<<<(num_node_arr[i+1] + 1023) / 1024, 1024>>>(layer[i+1].a, num_node_arr[i+1]);
			}
			MatrixMultiply(layer[num_layer - 1].weight, layer[num_layer - 1].a, layer[num_layer].a, num_node_arr[num_layer], num_node_arr[num_layer - 1], 1);
			Exponential<<<(num_node_arr[num_layer] + 1023) / 1024, 1024>>>(layer[num_layer].a, num_node_arr[num_layer]);
			cublasDasum(handle, num_node_arr[num_layer], layer[num_layer].a, 1, &sum);
			Softmax<<<(num_node_arr[num_layer] + 1023) / 1024, 1024>>>(layer[num_layer].a, sum, num_node_arr[num_layer]);
			//Print<<<1, 1>>>(layer[num_layer].a, num_node_arr[num_layer]);
			//cudaDeviceSynchronize();
			
			cublasIdamax(handle, num_node_arr[num_layer], layer[num_layer].a, 1, &max_index);
			if (result_index == --max_index)
				correct++;
			num_test_examples++;
		}
	}
	if(pFile != NULL)
		fclose(pFile);
	
	printf("%lf%%\n", correct / num_test_examples * 100);
	return;
}*/

void MatrixMultiply(double *d_A, double *d_B, double *d_C, int A_H, int A_W, int B_W, int i)
{
	const double alp = 1.0f;
	const double bet  = 0.0f;
		
	hipblasDgemm(handle[i], HIPBLAS_OP_N, HIPBLAS_OP_N, B_W, A_H, A_W, &alp, d_B, B_W, d_A, A_W, &bet, d_C, B_W);
}

void InputForwardComputation(int index)
{
	int buffer = index % 2;
	hipSetDevice(0);
	
	// copy input data
	hipMemcpyAsync(layer[0].a[buffer], input_d + (index * num_node_arr[0]) * sizeof(int), num_node_arr[0] * sizeof(double), hipMemcpyDeviceToDevice, stream[0]);
	
	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[0]>>>(layer[0].is_fw_output_ready, buffer, __LINE__, index);
	
	// compute activation
	MatrixMultiply(layer[0].a[buffer], layer[0].weight, layer[0].a_next[buffer], 1, num_node_arr[0], num_node_arr[1], 0);
	Sigmoid<<<(num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a_next[buffer], num_node_arr[1]);
	
	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[0].is_fw_output_ready[buffer], 1, sizeof(int), stream[0]);
}

void InputForwardCommunication(int index)
{
	int buffer = index % 2;
	hipSetDevice(0);

	// wait for current layer's forward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[1]>>>(layer[0].is_fw_output_ready, buffer, __LINE__, index);
	
	// wait for next layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[1]>>>(layer[0].is_fw_next_input_ready, buffer, __LINE__, index);

	// copy foward output (activation) to next layer
	hipMemcpyPeerAsync(layer[1].a[buffer], 1, layer[0].a_next[buffer], 0, num_node_arr[1] * sizeof(double), stream[1]);
	
	// next layer's forward input buffer is full
	hipMemsetAsync(&layer[1].is_fw_input_ready[buffer], 1, sizeof(int), stream[1]);
	hipMemsetAsync(&layer[0].is_fw_next_input_ready[buffer], 1, sizeof(int), stream[1]);

	// current layer's forward output buffer is empty
	hipMemsetAsync(&layer[0].is_fw_output_ready[buffer], 0, sizeof(int), stream[1]);
}

void InputForwardBackwardComputation(int fw_index, int bw_index)
{
	int fw_buffer = fw_index % 2;
	int bw_buffer = bw_index % 2;
	hipSetDevice(0);

	// copy input data
	hipMemcpyAsync(layer[0].a[fw_buffer], input_d + (fw_index * num_node_arr[0]) * sizeof(int), num_node_arr[0] * sizeof(double), hipMemcpyDeviceToDevice, stream[0]);
	
	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[0]>>>(layer[0].is_fw_output_ready, fw_buffer, __LINE__, fw_index);
	
	// compute activation
	MatrixMultiply(layer[0].a[fw_buffer], layer[0].weight, layer[0].a_next[fw_buffer], 1, num_node_arr[0], num_node_arr[1], 0);
	Sigmoid<<<(num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a_next[fw_buffer], num_node_arr[1]);
	
	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[0].is_fw_output_ready[fw_buffer], 1, sizeof(int), stream[0]);

	// wait for current layer's backward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[0]>>>(layer[0].is_bw_input_ready, bw_buffer, __LINE__, bw_index);

	// update weight
	UpdateWeight<<<(num_node_arr[0] * num_node_arr[1] + 1023) / 1024, 1024, 0, stream[0]>>>(layer[0].a[bw_buffer], layer[0].weight, layer[0].delta_next[bw_buffer], num_node_arr[1], num_node_arr[1] * num_node_arr[0], alpha);

	// current layer's backward input buffer is empty
	hipMemsetAsync(&layer[0].is_bw_input_ready[bw_buffer], 0, sizeof(int), stream[0]);
	hipMemsetAsync(&layer[1].is_bw_prev_input_ready[bw_buffer], 0, sizeof(int), stream[0]);
}

void HiddenForwardComputation(int device, int index)
{
	int buffer = index % 2;
	hipSetDevice(device);

	// wait for current layer's forward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_input_ready, buffer, __LINE__, index);

	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_output_ready, buffer, __LINE__, index);

	// compute activation
	MatrixMultiply(layer[device].a[buffer], layer[device].weight, layer[device].a_next[buffer], 1, num_node_arr[device], num_node_arr[device + 1], device);
	if(device != num_layer - 2)
		Sigmoid<<<(num_node_arr[device + 1] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].a_next[buffer], num_node_arr[device + 1]);

	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[device].is_fw_output_ready[buffer], 1, sizeof(int), stream[2 * device]);

	// current layer's forward input buffer is empty
	hipMemsetAsync(&layer[device].is_fw_input_ready[buffer], 0, sizeof(int), stream[2 * device]);
	hipMemsetAsync(&layer[device - 1].is_fw_next_input_ready[buffer], 0, sizeof(int), stream[2 * device]);
}

void HiddenForwardCommunication(int device, int index)
{
	int buffer = index % 2;
	hipSetDevice(device);
	
	// wait for current layer's forward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_output_ready, buffer, __LINE__, index);
				
	// wait for next layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_next_input_ready, buffer, __LINE__, index);

	// copy forward output (activation) to next layer
	hipMemcpyPeerAsync(layer[device + 1].a[buffer], device + 1, layer[device].a_next[buffer], device, num_node_arr[device + 1] * sizeof(double), stream[2 * device + 1]);

	// next layer's forward input buffer is full
	hipMemsetAsync(&layer[device + 1].is_fw_input_ready[buffer], 1, sizeof(int), stream[2 * device + 1]);
	hipMemsetAsync(&layer[device].is_fw_next_input_ready[buffer], 1, sizeof(int), stream[2 * device + 1]);

	// current layer's forward output buffer is empty
	hipMemsetAsync(&layer[device].is_fw_output_ready[buffer], 0, sizeof(int), stream[2 * device + 1]);
}

void HiddenForwardBackwardComputation(int device, int fw_index, int bw_index)
{
	int fw_buffer = fw_index % 2;
	int bw_buffer = bw_index % 2;
	hipSetDevice(device);

	// wait for current layer's forward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_input_ready, fw_buffer, __LINE__, fw_index);

	// wait for current layer's forward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_fw_output_ready, fw_buffer, __LINE__, fw_index);

	// compute activation
	MatrixMultiply(layer[device].a[fw_buffer], layer[device].weight, layer[device].a_next[fw_buffer], 1, num_node_arr[device], num_node_arr[device + 1], device);
	if(device != num_layer - 2)
		Sigmoid<<<(num_node_arr[device + 1] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].a_next[fw_buffer], num_node_arr[device + 1]);

	// current layer's forward output buffer is full
	hipMemsetAsync(&layer[device].is_fw_output_ready[fw_buffer], 1, sizeof(int), stream[2 * device]);

	// wait for current layer's backward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_bw_input_ready, bw_buffer, __LINE__, bw_index);

	// wait for current layer's backward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device]>>>(layer[device].is_bw_output_ready, bw_buffer, __LINE__, bw_index);

	// compute backward output (delta)
	MatrixMultiply(layer[device].weight, layer[device].delta_next[bw_buffer], layer[device].delta[bw_buffer], num_node_arr[device], num_node_arr[device + 1], 1, device);
	GetHiddenLayerDelta<<<(num_node_arr[device] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].delta[bw_buffer], layer[device].a[bw_buffer], layer[device].weight, layer[device].delta_next[bw_buffer], num_node_arr[device]);

	// current layer's backward output buffer is full
	hipMemsetAsync(&layer[device].is_bw_output_ready[bw_buffer], 1, sizeof(int), stream[2 * device]);

	// update weight
	UpdateWeight<<<(num_node_arr[device] * num_node_arr[device + 1] + 1023) / 1024, 1024, 0, stream[2 * device]>>>(layer[device].a[bw_buffer], layer[device].weight, layer[device].delta_next[bw_buffer], num_node_arr[device + 1], num_node_arr[device + 1] * num_node_arr[device], alpha);

	// current layer's forward input buffer is empty
	hipMemsetAsync(&layer[device].is_fw_input_ready[fw_buffer], 0, sizeof(int), stream[2 * device]);
	hipMemsetAsync(&layer[device - 1].is_fw_next_input_ready[fw_buffer], 0, sizeof(int), stream[2 * device]);

	// current layer's backward input buffer is empty
	hipMemsetAsync(&layer[device].is_bw_input_ready[bw_buffer], 0, sizeof(int), stream[2 * device]);
	hipMemsetAsync(&layer[device + 1].is_bw_prev_input_ready[bw_buffer], 0, sizeof(int), stream[2 * device]);
}

void HiddenForwardBackwardCommunication(int device, int fw_index, int bw_index)
{
	int fw_buffer = fw_index % 2;
	int bw_buffer = bw_index % 2;
	hipSetDevice(device);

	// wait for current layer's forward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_output_ready, fw_buffer, __LINE__, fw_index);
				
	// wait for next layer's forward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_fw_next_input_ready, fw_buffer, __LINE__, fw_index);

	// copy forward output (activation) to next layer
	hipMemcpyPeerAsync(layer[device + 1].a[fw_buffer], device + 1, layer[device].a_next[fw_buffer], device, num_node_arr[device + 1] * sizeof(double), stream[2 * device + 1]);

	// next layer's forward input buffer is full
	hipMemsetAsync(&layer[device + 1].is_fw_input_ready[fw_buffer], 1, sizeof(int), stream[2 * device + 1]);
	hipMemsetAsync(&layer[device].is_fw_next_input_ready[fw_buffer], 1, sizeof(int), stream[2 * device + 1]);

	// current layer's forward output buffer is empty
	hipMemsetAsync(&layer[device].is_fw_output_ready[fw_buffer], 0, sizeof(int), stream[2 * device + 1]);

	// wait for current layer's backward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_bw_output_ready, bw_buffer, __LINE__, bw_index);

	// wait for previous layer's backward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * device + 1]>>>(layer[device].is_bw_prev_input_ready, bw_buffer, __LINE__, bw_index);

	// copy backward output (delta) to preious layer
	hipMemcpyPeerAsync(layer[device - 1].delta_next[bw_buffer], device - 1, layer[device].delta[bw_buffer], device, num_node_arr[device - 1] * sizeof(double), stream[2 * device + 1]);

	// previous layer's backward input buffer is full
	hipMemsetAsync(&layer[device - 1].is_bw_input_ready[bw_buffer], 1, sizeof(int), stream[2 * device + 1]);
	hipMemsetAsync(&layer[device].is_bw_prev_input_ready[bw_buffer], 1, sizeof(int), stream[2 * device + 1]);

	// current layer's backward output buffer is empty
	hipMemsetAsync(&layer[device].is_bw_output_ready[bw_buffer], 0, sizeof(int), stream[2 * device + 1]);
}

void OutputForwardBackwardComputation(int fw_index, int bw_index)
{
	int fw_buffer = fw_index % 2;
	int bw_buffer = bw_index % 2;
	hipSetDevice(num_layer - 1);

	// wait for current layer's forward input buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].is_fw_input_ready, fw_buffer, __LINE__, fw_index);

	// softmax
	Exponential<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[fw_buffer], num_node_arr[num_layer - 1]);
	hipblasDasum(handle[num_layer - 1], num_node_arr[num_layer - 1] - 1, layer[num_layer - 1].a[fw_buffer], 1, &sum[fw_buffer]);
	Softmax<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[fw_buffer], sum[fw_buffer], num_node_arr[num_layer - 1]);

	// wait for current layer's backward output buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].is_bw_output_ready, bw_buffer, __LINE__, bw_index);
		
	// compute delta
	GetOutputLayerDelta<<<(num_node_arr[num_layer - 1] + 1023) / 1024, 1024, 0, stream[2 * (num_layer - 1)]>>>(layer[num_layer - 1].a[bw_buffer], layer[num_layer - 1].delta[bw_buffer], result_d, bw_index, num_node_arr[num_layer - 1]);

	// current layer's backward output buffer is full
	hipMemsetAsync(&layer[num_layer - 1].is_bw_output_ready[bw_buffer], 1, sizeof(int), stream[2 * (num_layer - 1)]);

	// current layer's forward input buffer is empty
	hipMemsetAsync(&layer[num_layer - 1].is_fw_input_ready[fw_buffer], 0, sizeof(int), stream[2 * (num_layer - 1)]);
	hipMemsetAsync(&layer[num_layer - 2].is_fw_next_input_ready[fw_buffer], 0, sizeof(int), stream[2 * (num_layer - 1)]);
}

void OutputForwardBackwardCommunication(int fw_index, int bw_index)
{
	int bw_buffer = bw_index % 2;
	hipSetDevice(num_layer - 1);

	// wait for previous layer's backward input buffer is empty
	WaituntilZero<<<1, 1, 0, stream[2 * num_layer - 1]>>>(layer[num_layer - 1].is_bw_prev_input_ready, bw_buffer, __LINE__, bw_index);

	// wait for current layer's backward output buffer is full
	WaituntilOne<<<1, 1, 0, stream[2 * num_layer - 1]>>>(layer[num_layer - 1].is_bw_output_ready, bw_buffer, __LINE__, bw_index);

	// copy backward output (delta) to previous layer
	hipMemcpyPeerAsync(layer[num_layer - 2].delta_next[bw_buffer], num_layer - 2, layer[num_layer - 1].delta[bw_buffer], num_layer - 1, num_node_arr[num_layer - 1] * sizeof(double), stream[2 * num_layer -1]);

	// previous layer's backward input buffer is full
	hipMemsetAsync(&layer[num_layer - 2].is_bw_input_ready[bw_buffer], 1, sizeof(int), stream[2 * num_layer - 1]);
	hipMemsetAsync(&layer[num_layer - 1].is_bw_prev_input_ready[bw_buffer], 1, sizeof(int), stream[2 * num_layer - 1]);

	// current layer's backward output buffer is empty
	hipMemsetAsync(&layer[num_layer - 1].is_bw_output_ready[bw_buffer], 0, sizeof(int), stream[2 * num_layer - 1]);
}

__global__ void GetOutputLayerDelta(double *output_a, double *output_delta, double *result, int index, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
	{
		if(i != result[index])
			output_delta[i] = 0.0 - output_a[i];
		else
			output_delta[i] = 1.0 - output_a[i];
	}
}

__global__ void GetHiddenLayerDelta(double *cur_delta, double *cur_a, double *cur_weight, double *prev_delta, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		cur_delta[i] = (1.0 - cur_a[i]) * cur_a[i] * (cur_delta[i] - cur_weight[i] * prev_delta[0]);
}

__global__ void UpdateWeight(double *cur_a, double *cur_weight, double *next_delta, int W_W, int size, double lr)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		cur_weight[i] = cur_weight[i] + lr * cur_a[i / W_W] * next_delta[i % W_W];
}

__global__ void Sigmoid(double *a, int size) 
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = 1.0 / (1.0 + exp(-a[i]));
	if(i == 0)
		a[i] = 1.0;
}

__global__ void Exponential(double *a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] = exp(a[i]);
	if(i == 0)
		a[i] = 0.0;
}

__global__ void Softmax(double *a, double sum, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < size)
		a[i] /= sum;
}

__global__ void WaituntilZero(int *ready, int index, int line, int x)
{
	while(ready[index] == 1)
	{
		printf("%d\t%d\n", line, x);
	}
}

__global__ void WaituntilOne(int *ready, int index, int line, int x)
{
	while(ready[index] == 0)
	{
		printf("%d\t%d\n", line, x);
	}
}

__global__ void SetFlag(int *ready, int index)
{
	ready[index] = 1 - ready[index];
}

__global__ void PrintFw(int device, int index)
{
	printf("device #%d\t fw -> %d\n", device, index);
}

__global__ void PrintBw(int device, int index)
{
	printf("device #%d\t bw -> %d\n", device, index);
}

__global__ void PrintInt(int *flag, int index, int line)
{
	printf("line #%d\t index #%d\t %d\n", line, index, flag[index]);
}

__global__ void PrintDouble(double *flag, int index, int size, int line)
{
	printf("line #%d\t", line);
	for(int i = 0; i < size; i++)
		printf("%lf ", flag[index + i]);
	printf("\n");
}
