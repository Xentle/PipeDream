
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dummy()
{
	int j = 0;
	for(int i = 0; i < 1000000; i++)
		j++;
}

int main()
{
	hipStream_t stream1, stream2;

	double *A, *B, *C, *D;

	hipSetDevice(2);
	hipMalloc((void **) &C, 100000000 * sizeof(double));

	hipSetDevice(0);
	hipMalloc((void **) &D, 100000000 * sizeof(double));

	hipSetDevice(1);
	hipMalloc((void **) &A, 100000000 * sizeof(double));
	hipMalloc((void **) &B, 100000000 * sizeof(double));

	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipDeviceEnablePeerAccess(2, 0);
	hipDeviceEnablePeerAccess(0, 0);

	hipMemcpyPeerAsync(C, 2, A, 1, 100000000 * sizeof(double), stream1);
	hipMemcpyPeerAsync(D, 0, B, 1, 100000000 * sizeof(double), stream2);

	for(int i = 0; i < 3; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}
	return 0;
}

