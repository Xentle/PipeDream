
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dummy()
{
	int j = 0;
	for(int i = 0; i < 1000000; i++)
		j++;
}

int main()
{
	hipStream_t stream1, stream2;

	double *A, *B, *C, *D;

	hipSetDevice(1);
	hipMalloc((void **) &C, 100000000 * sizeof(double));
	hipMalloc((void **) &D, 10000000 * sizeof(double));

	hipSetDevice(0);
	hipMalloc((void **) &A, 100000000 * sizeof(double));
	hipMalloc((void **) &B, 10000000 * sizeof(double));

	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipDeviceEnablePeerAccess(1, 0);

	dummy<<<1, 1>>>();

	hipSetDevice(0);
	hipMemcpyPeerAsync(C, 1, A, 0, 100000000 * sizeof(double), stream1);

	hipSetDevice(1);
	for(int i = 0; i < 10; i++)
		dummy<<<1, 1>>>();

	hipSetDevice(0);
	hipMemcpyPeerAsync(D, 1, B, 0, 10000000 * sizeof(double));

	
	
	for(int i = 0; i < 2; i++)
	{
		hipSetDevice(i);
		hipDeviceSynchronize();
	}
	return 0;
}

